#include "hip/hip_runtime.h"
#include "CurvePerturbUtils.h"

#include "PathWeightUtils.h"

#include <cmath>

// Cuda Functions
namespace twisty {
// Assumes pVector3f is an array of 3 floats
__host__ __device__ void NormalizeVector3f(float *pVector3f)
{
    float normalizer
          = pVector3f[0] * pVector3f[0] + pVector3f[1] * pVector3f[1] + pVector3f[2] * pVector3f[2];
    normalizer = 1.0f / std::sqrt(normalizer);
    pVector3f[0] *= normalizer;
    pVector3f[1] *= normalizer;
    pVector3f[2] *= normalizer;
}

// This has an outparameter
__host__ __device__ void RotationMatrixAroundAxis(
      const float angle, const float *pAxisVector3f, float *pMatrix3x3)
{
    // Ensure its normalized
    // TODO: Make assertion
    // NormalizeVector3f(pAxisVector3f);

    pMatrix3x3[0]
          = std::cos(angle) + pAxisVector3f[0] * pAxisVector3f[0] * (1.0f - std::cos(angle));
    pMatrix3x3[1] = pAxisVector3f[0] * pAxisVector3f[1] * (1.0f - std::cos(angle))
          - pAxisVector3f[2] * std::sin(angle);
    pMatrix3x3[2] = pAxisVector3f[0] * pAxisVector3f[2] * (1.0f - std::cos(angle))
          + pAxisVector3f[1] * std::sin(angle);

    pMatrix3x3[3] = pAxisVector3f[1] * pAxisVector3f[0] * (1.0f - std::cos(angle))
          + pAxisVector3f[2] * std::sin(angle);
    pMatrix3x3[4] = std::cos(angle) + pAxisVector3f[1] * pAxisVector3f[1] * (1 - std::cos(angle));
    pMatrix3x3[5] = pAxisVector3f[1] * pAxisVector3f[2] * (1 - std::cos(angle))
          - pAxisVector3f[0] * std::sin(angle);

    pMatrix3x3[6] = pAxisVector3f[2] * pAxisVector3f[0] * (1 - std::cos(angle))
          - pAxisVector3f[1] * std::sin(angle);
    pMatrix3x3[7] = pAxisVector3f[2] * pAxisVector3f[1] * (1 - std::cos(angle))
          + pAxisVector3f[0] * std::sin(angle);
    pMatrix3x3[8] = std::cos(angle) + pAxisVector3f[2] * pAxisVector3f[2] * (1 - std::cos(angle));
}

__host__ __device__ float DotVector3fVector3f(float *lhs, float *rhs)
{
    return lhs[0] * rhs[0] + lhs[1] * rhs[1] + lhs[2] * rhs[2];
}

__host__ __device__ float MagVector3f(float *pVec)
{
    return sqrt(pVec[0] * pVec[0] + pVec[1] * pVec[1] + pVec[2] * pVec[2]);
}

__host__ __device__ void RotateVectorByMatrix(float *pRotationMatrix, float *pVector)
{
    float val[3];
    val[0] = DotVector3fVector3f(pRotationMatrix, pVector);
    val[1] = DotVector3fVector3f(pRotationMatrix + 3, pVector);
    val[2] = DotVector3fVector3f(pRotationMatrix + 6, pVector);

    // Write it back to pVector
    pVector[0] = val[0];
    pVector[1] = val[1];
    pVector[2] = val[2];
}
}


namespace twisty {
namespace PerturbUtils {
    // This function assumes that the initial and end positions and tangents are set already to the constraints defined by the problem
    __host__ void UpdateTangentsFromPos(Farlor::Vector3 *pPositions, Farlor::Vector3 *pTangents,
          const uint32_t numSegments, const BoundaryConditions &boundaryConditions)
    {
        BoundaryConditions_CudaSafe cs;
        cs.m_startPos[0] = boundaryConditions.m_startPos.m_data[0];
        cs.m_startPos[1] = boundaryConditions.m_startPos.m_data[1];
        cs.m_startPos[2] = boundaryConditions.m_startPos.m_data[2];

        cs.m_startDir[0] = boundaryConditions.m_startDir.m_data[0];
        cs.m_startDir[1] = boundaryConditions.m_startDir.m_data[1];
        cs.m_startDir[2] = boundaryConditions.m_startDir.m_data[2];

        cs.m_endPos[0] = boundaryConditions.m_endPos.m_data[0];
        cs.m_endPos[1] = boundaryConditions.m_endPos.m_data[1];
        cs.m_endPos[2] = boundaryConditions.m_endPos.m_data[2];

        cs.m_endDir[0] = boundaryConditions.m_endDir.m_data[0];
        cs.m_endDir[1] = boundaryConditions.m_endDir.m_data[1];
        cs.m_endDir[2] = boundaryConditions.m_endDir.m_data[2];

        cs.arclength = boundaryConditions.arclength;

        UpdateTangentsFromPosCudaSafe((float *)pPositions, (float *)pTangents, numSegments, cs);
    }

    // This function assumes that the initial and end positions and tangents are set already to the constraints defined by the problem
    __host__ void UpdateCurvaturesFromTangents(Farlor::Vector3 *pTangents, float *pCurvatures,
          const uint32_t numSegments, const BoundaryConditions &boundaryConditions,
          int32_t weightingMethod)
    {
        BoundaryConditions_CudaSafe cs;
        cs.m_startPos[0] = boundaryConditions.m_startPos.m_data[0];
        cs.m_startPos[1] = boundaryConditions.m_startPos.m_data[1];
        cs.m_startPos[2] = boundaryConditions.m_startPos.m_data[2];

        cs.m_startDir[0] = boundaryConditions.m_startDir.m_data[0];
        cs.m_startDir[1] = boundaryConditions.m_startDir.m_data[1];
        cs.m_startDir[2] = boundaryConditions.m_startDir.m_data[2];

        cs.m_endPos[0] = boundaryConditions.m_endPos.m_data[0];
        cs.m_endPos[1] = boundaryConditions.m_endPos.m_data[1];
        cs.m_endPos[2] = boundaryConditions.m_endPos.m_data[2];

        cs.m_endDir[0] = boundaryConditions.m_endDir.m_data[0];
        cs.m_endDir[1] = boundaryConditions.m_endDir.m_data[1];
        cs.m_endDir[2] = boundaryConditions.m_endDir.m_data[2];

        cs.arclength = boundaryConditions.arclength;

        UpdateCurvaturesFromTangentsCudaSafe(
              (float *)pTangents, pCurvatures, numSegments, cs, weightingMethod);
    }

    // This function assumes that the initial and end positions and tangents are set already to the constraints defined by the problem
    __host__ __device__ void UpdateTangentsFromPosCudaSafe(float *pPositions, float *pTangents,
          const uint32_t numSegments, const BoundaryConditions_CudaSafe &csBoundaryConditions)
    {
        const float invDS = numSegments / csBoundaryConditions.arclength;

        // TODO: Unnecessary operations, remove?
        // Set initial and final positions
        //pPositions[0 * 3 + 0] = csBoundaryConditions.m_startPos[0];
        //pPositions[0 * 3 + 1] = csBoundaryConditions.m_startPos[1];
        //pPositions[0 * 3 + 2] = csBoundaryConditions.m_startPos[2];

        //pPositions[1 * 3 + 0] = pPositions[0 * 3 + 0] + ds * csBoundaryConditions.m_startDir[0];
        //pPositions[1 * 3 + 1] = pPositions[0 * 3 + 1] + ds * csBoundaryConditions.m_startDir[1];
        //pPositions[1 * 3 + 2] = pPositions[0 * 3 + 2] + ds * csBoundaryConditions.m_startDir[2];

        //pPositions[numSegments * 3 + 0] = csBoundaryConditions.m_endPos[0];
        //pPositions[numSegments * 3 + 1] = csBoundaryConditions.m_endPos[1];
        //pPositions[numSegments * 3 + 2] = csBoundaryConditions.m_endPos[2];

        // TODO: Is Forward Difference good enough?
        for (uint32_t i = 0; i < numSegments; ++i) {
            float diff_x = pPositions[((i + 1) * 3) + 0] - pPositions[(i * 3) + 0];
            float diff_y = pPositions[((i + 1) * 3) + 1] - pPositions[(i * 3) + 1];
            float diff_z = pPositions[((i + 1) * 3) + 2] - pPositions[(i * 3) + 2];
            pTangents[i * 3 + 0] = diff_x * invDS;
            pTangents[i * 3 + 1] = diff_y * invDS;
            pTangents[i * 3 + 2] = diff_z * invDS;

            //TODO: Should we normalize the damn tangents?
            float mag = pTangents[i * 3 + 0] * pTangents[i * 3 + 0]
                  + pTangents[i * 3 + 1] * pTangents[i * 3 + 1]
                  + pTangents[i * 3 + 2] * pTangents[i * 3 + 2];
            mag = std::sqrt(mag);
            pTangents[i * 3 + 0] /= mag;
            pTangents[i * 3 + 1] /= mag;
            pTangents[i * 3 + 2] /= mag;
        }
    }

    // This function assumes that the initial and end positions and tangents are set already to the constraints defined by the problem
    __host__ __device__ void UpdateCurvaturesFromTangentsCudaSafe(float *pTangents,
          float *pCurvatures, const uint32_t numSegments,
          const BoundaryConditions_CudaSafe &boundaryConditions, int32_t weightingMethod)
    {
        const float invDs = numSegments / boundaryConditions.arclength;

        switch (weightingMethod) {
            case (int32_t)twisty::WeightingMethod::RadiativeTransfer: {
                // Update segments
                for (uint32_t i = 0; i < (numSegments - 1); ++i) {
                    const float tanLeft_x = pTangents[i * 3 + 0];
                    const float tanLeft_y = pTangents[i * 3 + 1];
                    const float tanLeft_z = pTangents[i * 3 + 2];

                    const float tanRight_x = pTangents[(i + 1) * 3 + 0];
                    const float tanRight_y = pTangents[(i + 1) * 3 + 1];
                    const float tanRight_z = pTangents[(i + 1) * 3 + 2];

                    {
                        const float scaledDiff_x = (tanRight_x - tanLeft_x) * invDs;
                        const float scaledDiff_y = (tanRight_y - tanLeft_y) * invDs;
                        const float scaledDiff_z = (tanRight_z - tanLeft_z) * invDs;

                        pCurvatures[i] = sqrt(scaledDiff_x * scaledDiff_x
                              + scaledDiff_y * scaledDiff_y + scaledDiff_z * scaledDiff_z);
                    }
                }
            } break;
            case (int32_t)twisty::WeightingMethod::SimplifiedModel: {
                // Update segments
                for (uint32_t i = 0; i < (numSegments - 1); ++i) {
                    float tanLeft_x = pTangents[i * 3 + 0];
                    float tanLeft_y = pTangents[i * 3 + 1];
                    float tanLeft_z = pTangents[i * 3 + 2];
                    const float leftLength = sqrt(
                          tanLeft_x * tanLeft_x + tanLeft_y * tanLeft_y + tanLeft_z * tanLeft_z);
                    tanLeft_x /= leftLength;
                    tanLeft_y /= leftLength;
                    tanLeft_z /= leftLength;

                    float tanRight_x = pTangents[(i + 1) * 3 + 0];
                    float tanRight_y = pTangents[(i + 1) * 3 + 1];
                    float tanRight_z = pTangents[(i + 1) * 3 + 2];
                    const float rightLength = sqrt(tanRight_x * tanRight_x + tanRight_y * tanRight_y
                          + tanRight_z * tanRight_z);
                    tanRight_x /= rightLength;
                    tanRight_y /= rightLength;
                    tanRight_z /= rightLength;

                    {
                        const float curvature = (tanLeft_x * tanRight_x) + (tanLeft_y * tanRight_y)
                              + (tanLeft_z * tanRight_z);
                        pCurvatures[i]
                              = -curvature;  // Negate curvature so that an increase in curvature leads to higher weight values
                    }
                }
            } break;
            default: {
            } break;
        };
    }
}
}