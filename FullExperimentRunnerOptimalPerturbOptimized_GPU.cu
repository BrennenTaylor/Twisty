#include "hip/hip_runtime.h"
#include "FullExperimentRunnerOptimalPerturbOptimized_GPU.h"

#include <boost\multiprecision\cpp_dec_float.hpp>

#include "CurvePerturbUtils.h"
#include "CurveUtils.h"
#include "MathConsts.h"
#include "PathWeightUtils.h"

#include <hiprand.h>

#include <omp.h>

#include <assert.h>
#include <ctime>
#include <fstream>
#include <filesystem>
#include <limits>

#include <chrono>
#include <thread>
#include <mutex>
#include <condition_variable>
#include <atomic>
#include <stdlib.h>
#include <memory>

const uint32_t PerturbWarpSize = 32;
const uint32_t PerturbGridSize = 10;

namespace twisty
{
    static void CudaSafeErrorCheck(hipError_t error, std::string message)
    {
        if (error != hipSuccess)
        {
            std::string errorString(hipGetErrorString(error));
            fprintf(stderr, "ERROR: %s : %s\n", message.c_str(), errorString.c_str());
            assert(false);
        }
    }

            // Assumes pVector3f is an array of 3 floats
    static __host__ __device__ void NormalizeVector3f(float* pVector3f)
    {
        float normalizer = pVector3f[0] * pVector3f[0] + pVector3f[1] * pVector3f[1] + pVector3f[2] * pVector3f[2];
        normalizer = 1.0f / sqrt(normalizer);
        pVector3f[0] *= normalizer;
        pVector3f[1] *= normalizer;
        pVector3f[2] *= normalizer;
    }

    // This has an out parameter
    static __host__ __device__  void RotationMatrixAroundAxis(float angle, float* pAxisVector3f, float* pMatrix3x3)
    {
        // Ensure its normalized
        NormalizeVector3f(pAxisVector3f);

        pMatrix3x3[0] = cos(angle) + pAxisVector3f[0] * pAxisVector3f[0] * (1.0f - cos(angle));
        pMatrix3x3[1] = pAxisVector3f[0] * pAxisVector3f[1] * (1.0f - cos(angle)) - pAxisVector3f[2] * sin(angle);
        pMatrix3x3[2] = pAxisVector3f[0] * pAxisVector3f[2] * (1.0f - cos(angle)) + pAxisVector3f[1] * sin(angle);

        pMatrix3x3[3] = pAxisVector3f[1] * pAxisVector3f[0] * (1.0f - cos(angle)) + pAxisVector3f[2] * sin(angle);
        pMatrix3x3[4] = cos(angle) + pAxisVector3f[1] * pAxisVector3f[1] * (1 - cos(angle));
        pMatrix3x3[5] = pAxisVector3f[1] * pAxisVector3f[2] * (1 - cos(angle)) - pAxisVector3f[0] * sin(angle);

        pMatrix3x3[6] = pAxisVector3f[2] * pAxisVector3f[0] * (1 - cos(angle)) - pAxisVector3f[1] * sin(angle);
        pMatrix3x3[7] = pAxisVector3f[2] * pAxisVector3f[1] * (1 - cos(angle)) + pAxisVector3f[0] * sin(angle);
        pMatrix3x3[8] = cos(angle) + pAxisVector3f[2] * pAxisVector3f[2] * (1 - cos(angle));
    }

    static __host__ __device__  float DotVector3fVector3f(float* lhs, float* rhs)
    {
        return lhs[0] * rhs[0] + lhs[1] * rhs[1] + lhs[2] * rhs[2];
    }

    // TODO: Check if needed during rewrite
    // static __host__ __device__  float MagVector3f(float* pVec)
    // {
    //     return sqrt(pVec[0] * pVec[0] + pVec[1] * pVec[1] + pVec[2] * pVec[2]);
    // }

    static __host__ __device__  void RotateVectorByMatrix(float* pRotationMatrix, float* pVector)
    {
        float val[3];
        val[0] = DotVector3fVector3f(pRotationMatrix, pVector);
        val[1] = DotVector3fVector3f(pRotationMatrix + 3, pVector);
        val[2] = DotVector3fVector3f(pRotationMatrix + 6, pVector);
        
        // Write it back to pVector
        pVector[0] = val[0];
        pVector[1] = val[1];
        pVector[2] = val[2];
    }

        __global__ void FullExperimentRunnerOptimalPerturbOptimized_GPU_GeometryRandomKernel(
            int64_t numCombinedWeightValuesTotal,
            int64_t numCombinedWeightValuesPerWarp,
            int64_t numPathsPerThread,
            int64_t numPathsToSkipPerThread,
            int64_t numSegmentsPerCurve,
            hiprandState_t *pCurandStates,
            float *pPerGlobalThreadScratchSpacePositions,
            float *pPerGlobalThreadScratchSpaceTangents,
            float *pPerGlobalThreadScratchSpaceCurvatures,
            CombinedWeightValues_C *pPerThreadCombinedWeightValues,
            CombinedWeightValues_C *pFinalCombinedValues,
            const twisty::WeightingParameters &weightingParams,
            const twisty::PerturbUtils::BoundaryConditions_CudaSafe &csBoundaryConditions,
            const double *pLookupTable
        );

        FullExperimentRunnerOptimalPerturbOptimized_GPU::FullExperimentRunnerOptimalPerturbOptimized_GPU(ExperimentRunner::ExperimentParameters &experimentParams, Bootstrapper &bootstrapper)
        : ExperimentRunner(experimentParams, bootstrapper)
    {
    }

    FullExperimentRunnerOptimalPerturbOptimized_GPU::~FullExperimentRunnerOptimalPerturbOptimized_GPU()
    {
    }

    ExperimentRunner::RunnerSpecificResults FullExperimentRunnerOptimalPerturbOptimized_GPU::RunnerSpecificRunExperiment()
    {
        /* --------------------- */
        auto setupTimeStart = std::chrono::high_resolution_clock::now();
        /* --------------------- */


        // TODO: For now, we simply will support one scattering value 
        if (m_experimentParams.weightingParameters.scatterValues.size() > 1)
        {
            std::cout << "WARNING: Only one scatter value supported, defaulting to first specified scatter parameter" << std::endl;
        }
        m_experimentParams.weightingParameters.scatter = m_experimentParams.weightingParameters.scatterValues[0];
        std::unique_ptr<twisty::PathWeighting::BaseWeightLookupTable> lookupEvaluator = nullptr;

        if (m_experimentParams.weightingParameters.weightingMethod == WeightingMethod::SimplifiedModel)
        {
            lookupEvaluator = std::make_unique<twisty::PathWeighting::SimpleWeightLookupTable>(m_experimentParams.weightingParameters,
                                                                               m_upInitialCurve->m_segmentLength);
        }
        else
        {
            lookupEvaluator = std::make_unique<twisty::PathWeighting::WeightLookupTableIntegral>(m_experimentParams.weightingParameters,
                                                                                                 m_upInitialCurve->m_segmentLength);
        }
            
        lookupEvaluator->ExportValues(m_experimentParams.experimentDirPath);

        twisty::PerturbUtils::BoundaryConditions boundaryConditions = m_upInitialCurve->GetBoundaryConditions();

        bool result = SetupCudaDevice();
        if (!result)
        {
            printf("Failed to setup cuda device\n");
            return {};
        }

        // Calculate number of paths needed to generate

        const uint32_t warpPathCount = 1000000;
        const uint32_t numGlobalPerturbThreads = PerturbGridSize * PerturbWarpSize;

        const uint32_t numCombinedWeightValuesTotal = (m_experimentParams.numPathsInExperiment + MaxNumPathsPerCombinedWeight - 1) / MaxNumPathsPerCombinedWeight;
        const uint32_t numCombinedWeightValuesPerWarp = (numCombinedWeightValuesTotal + PerturbGridSize - 1) / PerturbGridSize;
        const uint32_t numPathsPerThread = (MaxNumPathsPerCombinedWeight + PerturbWarpSize - 1) / PerturbWarpSize;

        std::cout << "Num Global Perturb Threads: " << numGlobalPerturbThreads << std::endl;
        std::cout << "numPathsInExperiment: " << m_experimentParams.numPathsInExperiment << std::endl;
        std::cout << "numPathsPerBatch: " << warpPathCount << std::endl;
        std::cout << "Num Thread Batches: " << numCombinedWeightValuesTotal << std::endl;
        std::cout << "Num cached weights per warp: " << numCombinedWeightValuesPerWarp << std::endl;
        // std::cout << "Num paths per thread: " << numPathsPerThread << std::endl;
        std::cout << "Perturb Warp Size required: " << PerturbWarpSize << std::endl;
        std::cout << "Perturb Grid Size required: " << PerturbGridSize << std::endl;

        int32_t seed = m_experimentParams.curvePurturbSeed;
        if (seed == 0)
        {
            seed = time(0);
        }

        auto setupCuRandTimeStart = std::chrono::high_resolution_clock::now();
        {
            result = SetupCuRandStates(numGlobalPerturbThreads, seed);
            if (!result)
            {
                printf("Failed to setup CuRand states\n");
                return {};
            }
        }
        auto setupCuRandTimeEnd = std::chrono::high_resolution_clock::now();
 
        auto setupCudaPerturbStart = std::chrono::high_resolution_clock::now();
        {
            result = SetupCudaPerturb(numGlobalPerturbThreads, numCombinedWeightValuesTotal);
            if (!result)
            {
                printf("Failed to setup Cuda Perturb\n");
                return {};
            }
        }
        auto setupCudaPerturbEnd = std::chrono::high_resolution_clock::now();
        
        auto experimentTimeStart = std::chrono::high_resolution_clock::now();

        // Setup data structures
        std::vector<Farlor::Vector3> initialCurvePositions = m_upInitialCurve->m_positions;
        std::vector<Farlor::Vector3> initialCurveTangents(initialCurvePositions.size());
        std::vector<float> initialCurveCurvatures(initialCurvePositions.size() - 1);

        // Update and curvature
        twisty::PerturbUtils::UpdateTangentsFromPos(initialCurvePositions.data(), initialCurveTangents.data(),
            m_upInitialCurve->m_numSegments, boundaryConditions);
        twisty::PerturbUtils::UpdateCurvaturesFromTangents(initialCurveTangents.data(), initialCurveCurvatures.data(),
            m_upInitialCurve->m_numSegments, boundaryConditions, m_experimentParams.weightingParameters);

        const int64_t NumPosPerCurve = initialCurvePositions.size();
        const int64_t NumTanPerCurve = initialCurveTangents.size();
        const int64_t NumCurvaturePerCurve = initialCurveCurvatures.size();

        std::stringstream fnFilenameSS;
        fnFilenameSS << "SavedFN";
        fnFilenameSS << m_experimentParams.numSegmentsPerCurve;
        fnFilenameSS << ".fnd";
        const std::filesystem::path fnFilePath = std::filesystem::current_path() / fnFilenameSS.str();
        std::unique_ptr<PathWeighting::NormalizerStuff::BaseNormalizer> upFN = nullptr;

        // We dont need this actually, so we can just load the default one
        {
            // If we can load the fn data, load it
            if (std::filesystem::exists(fnFilePath))
            {
                std::cout << "Using cached fd file at: " << fnFilePath << std::endl;
                std::ifstream inFile(fnFilePath);
                upFN = std::make_unique<PathWeighting::NormalizerStuff::FN>(inFile);
                inFile.close();
            }
            // We need to generate it this time and save it off to use next time
            else
            {
                // This is the max M value
                const int maxorder = m_upInitialCurve->m_numSegments;

                // Generate the fn data table
                const int numZSamples = 5000;
                const int numIntegrationSamples = 5000;

                // Arbitrarily set min and max |r_vec| values.
                // Why this specific max bound, I do not know
                const double rMin = 0.0;
                const double rMax = 200.0;
                upFN = std::make_unique<PathWeighting::NormalizerStuff::FN>(numZSamples, numIntegrationSamples, maxorder, rMin, rMax);

                std::ofstream outFile(fnFilePath);
                dynamic_cast<PathWeighting::NormalizerStuff::FN*>(upFN.get())->WriteToFile(outFile);
                outFile.close();
            }
        }
        PathWeighting::NormalizerStuff::BaseNormalizer& fn = (*upFN);

        // Why the 1/(delta s) = (M+2)/s?
        Farlor::Vector3 Z = (boundaryConditions.m_endPos - boundaryConditions.m_startPos) * (m_upInitialCurve->m_numSegments + 2) / boundaryConditions.arclength
            - boundaryConditions.m_endDir - boundaryConditions.m_startDir;
        std::cout << "Z: " << Z << std::endl;
        std::cout << "|Z|: " << Z.Magnitude() << std::endl;

        PathWeighting::NormalizerStuff::NormalizerDoubleType pathNormalizer = 1.0;
        if (m_experimentParams.weightingParameters.weightingMethod == WeightingMethod::RadiativeTransfer)
        {
            pathNormalizer = PathWeighting::NormalizerStuff::Norm(fn, m_upInitialCurve->m_numSegments,
                                                 Z.Magnitude(), boundaryConditions.arclength);
        }
        const boost::multiprecision::cpp_dec_float_100 pathNormalizerLog10 = boost::multiprecision::log10(pathNormalizer);

        std::cout << "PathNormalizer: " << pathNormalizer << std::endl;
        std::cout << "PathNormalizerLog10: " << pathNormalizerLog10 << std::endl;

        auto setupTimeEnd = std::chrono::high_resolution_clock::now();
        /* --------------------- */


        /* --------------------- */

        long long perturbTimeCount = 0;
        long long weightCalcTimeCount = 0;

        std::cout << "numPathsInExperiment specified: " << m_experimentParams.numPathsInExperiment << std::endl;

        std::cout << "numPathsInExperiment generated: " << numCombinedWeightValuesTotal * MaxNumPathsPerCombinedWeight << std::endl;
        std::cout << "numCombinedWeightValuesTotal: " << numCombinedWeightValuesTotal << std::endl;
        std::cout << "numCombinedWeightValuesPerWarp: " << numCombinedWeightValuesPerWarp << std::endl;
        std::cout << "numPathsPerThread: " << numPathsPerThread << std::endl;

        std::vector<CombinedWeightValues_C> combinedWeightValues(numCombinedWeightValuesTotal);

        auto perturbTimeStart = std::chrono::high_resolution_clock::now();

        twisty::PerturbUtils::BoundaryConditions_CudaSafe csBoundaryConditions;
        csBoundaryConditions.m_startPos[0] = boundaryConditions.m_startPos[0];
        csBoundaryConditions.m_startPos[1] = boundaryConditions.m_startPos[1];
        csBoundaryConditions.m_startPos[2] = boundaryConditions.m_startPos[2];

        csBoundaryConditions.m_startDir[0] = boundaryConditions.m_startDir[0];
        csBoundaryConditions.m_startDir[1] = boundaryConditions.m_startDir[1];
        csBoundaryConditions.m_startDir[2] = boundaryConditions.m_startDir[2];

        csBoundaryConditions.m_endPos[0] = boundaryConditions.m_endPos[0];
        csBoundaryConditions.m_endPos[1] = boundaryConditions.m_endPos[1];
        csBoundaryConditions.m_endPos[2] = boundaryConditions.m_endPos[2];

        csBoundaryConditions.m_endDir[0] = boundaryConditions.m_endDir[0];
        csBoundaryConditions.m_endDir[1] = boundaryConditions.m_endDir[1];
        csBoundaryConditions.m_endDir[2] = boundaryConditions.m_endDir[2];

        csBoundaryConditions.arclength = boundaryConditions.arclength;

        {
            dim3 gridSize(PerturbWarpSize, 1, 1);
            dim3 blockSize(PerturbGridSize, 1, 1);
            size_t sharedMemorySizeBytes = 0;
            hipStream_t stream = 0;

            std::cout << "Dispatching with: " << std::endl;
            std::cout << "\tGrid Size: " << PerturbWarpSize << std::endl;
            std::cout << "\tBlock Size: " << PerturbGridSize << std::endl;

            FullExperimentRunnerOptimalPerturbOptimized_GPU_GeometryRandomKernel << <gridSize, blockSize, sharedMemorySizeBytes, stream >> >
                (
                    numCombinedWeightValuesTotal,
                    numCombinedWeightValuesPerWarp,
                    numPathsPerThread,
                    m_experimentParams.numPathsToSkip,
                    m_experimentParams.numSegmentsPerCurve,
                    m_pPerGlobalThreadRandStates,
                    m_pPerGlobalThreadScratchSpacePositions,
                    m_pPerGlobalThreadScratchSpaceTangents,
                    m_pPerGlobalThreadScratchSpaceCurvatures,
                    m_pPerThreadCombinedWeightValues,
                    m_pFinalCombinedValues,
                    m_experimentParams.weightingParameters,
                    csBoundaryConditions,
                    lookupEvaluator->AccessLookupTable().data()
                );

            CudaSafeErrorCheck(hipGetLastError(), "GPU_GeometryRandomKernel kernal launch");
            CudaSafeErrorCheck(hipDeviceSynchronize(), "GPU_GeometryRandomKernel kernel sync");
        }

        auto perturbTimeEnd = std::chrono::high_resolution_clock::now();
        perturbTimeCount += std::chrono::duration_cast<std::chrono::milliseconds>(perturbTimeEnd - perturbTimeStart).count();

        // -------------------
        auto weightingTimeStart = std::chrono::high_resolution_clock::now();

        // We need to calculate the absorbtion/scattering piece
        boost::multiprecision::cpp_dec_float_100 bigTotalExperimentWeight = 0.0;
        // No, we calculating the weighting
        for (auto& combinedWeightValue : combinedWeightValues)
        {
            bigTotalExperimentWeight += ExtractFinalValue(combinedWeightValue);
        }
        bigTotalExperimentWeight *= pathNormalizer;

        auto weightingTimeEnd = std::chrono::high_resolution_clock::now();
        weightCalcTimeCount += std::chrono::duration_cast<std::chrono::milliseconds>(weightingTimeEnd - weightingTimeStart).count();
        /* --------------------- */

        // Cleanup stuff

        {
            CleanupCudaPerturb();
            CleanupCudaRandStates();
            CleanupCudaDevice();
        }

        auto timeMs = std::chrono::duration_cast<std::chrono::milliseconds>(setupTimeEnd - setupTimeStart);

        ExperimentResults results;
        results.experimentWeights.push_back(bigTotalExperimentWeight);
        results.totalPathsGenerated = numCombinedWeightValuesTotal * MaxNumPathsPerCombinedWeight;
        results.numFailedPaths = 0;

        ExperimentRunner::RunnerSpecificResults specificResult;
        specificResult.experimentResults = std::make_optional<ExperimentResults>(results);
        specificResult.setupMsCount = timeMs.count();
        specificResult.runExperimentMsCount = perturbTimeCount;
        specificResult.weightingMsCount = weightCalcTimeCount;

        return specificResult;
    }

    bool FullExperimentRunnerOptimalPerturbOptimized_GPU::SetupCudaDevice()
    {
        // Query for the number of devices avalible
        int32_t numDevices = 0;
        CudaSafeErrorCheck(hipGetDeviceCount(&numDevices), "Get Devices");

        // We need at least one cuda device
        if (numDevices <= 0)
        {
            printf("No CUDA device avalible. Cannot execute experiment.\n");
            return false;
        }

        // Iterate over all devices and report the device stats.
        for (int32_t i = 0; i < numDevices; ++i)
        {
            hipDeviceProp_t prop;
            CudaSafeErrorCheck(hipGetDeviceProperties(&prop, i), "Get Device Prop");
            printf("\nDevice Number: %d\n", i);
            printf("\tDevice name: %s\n", prop.name);
            printf("\tSM Count: %d\n", prop.multiProcessorCount);
            printf("\tSM Shared Memory: %d\n", prop.sharedMemPerBlock);
            printf("\tWarp Size: %d\n", prop.warpSize);
            printf("\tThreads per SM: %d\n", prop.maxThreadsPerMultiProcessor);
            printf("\tPeak Memory Bandwidth (GB/s): %f\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
            printf("\tGlobal Memory: %zu\n", prop.totalGlobalMem);
            printf("\tConst Memory: %zu\n", prop.totalConstMem);
        }

        // We select the first device only
        const uint32_t selectedDeviceIdx = 0;
        std::cout << "\nSelected device: " << selectedDeviceIdx << std::endl;
        hipDeviceProp_t deviceProp;
        CudaSafeErrorCheck(hipGetDeviceProperties(&deviceProp, 0), "Get first device prop");

        m_numSMs = deviceProp.multiProcessorCount;
        m_warpSize = deviceProp.warpSize;
        m_maxThreadsPerMultiprocessor = deviceProp.maxThreadsPerMultiProcessor;

        return true;
    }

    void FullExperimentRunnerOptimalPerturbOptimized_GPU::CleanupCudaDevice()
    {
    }
    
    bool FullExperimentRunnerOptimalPerturbOptimized_GPU::SetupCuRandStates(uint32_t numGlobalPerturbThreads, uint32_t seed)
    {
        std::cout << "Setup Cuda Perturb: " << std::endl;
        uint64_t usedMemoryInBytes = 0;

        // Random Seed Kernel
        // Every block thread needs its own hiprand state
        CudaSafeErrorCheck(hipMalloc((void**)&m_pPerGlobalThreadRandStates, sizeof(hiprandState_t) * numGlobalPerturbThreads),
            "RandState malloc");
        usedMemoryInBytes += (sizeof(hiprandState_t) * numGlobalPerturbThreads);

        int blockSizeRandKernel = 0;
        int minGridSizeRandKernel = 0;
        size_t sharedMemoryUse = 0;
        size_t maxBlockSize = 0;
        CudaSafeErrorCheck(hipOccupancyMaxPotentialBlockSize(&minGridSizeRandKernel, &blockSizeRandKernel, FullExperimentRunnerOptimalPerturbOptimized_GPU_InitializeCurandState, sharedMemoryUse, maxBlockSize),
            "Failed to calculating occupancy for InitializeCuRandState kernel");
        std::cout << "\nInitializeCurandState: " << std::endl;
        std::cout << "\tBlock Size: " << blockSizeRandKernel << std::endl;
        std::cout << "\tMin Grid Size: " << minGridSizeRandKernel << std::endl;

        size_t gridSizeRandKernel = ((numGlobalPerturbThreads)+blockSizeRandKernel - 1) / blockSizeRandKernel;

        printf("\tInitializeCurandState Grid Size: %d\n", gridSizeRandKernel);
        printf("\tInitializeCurandState Block Size: %d\n", blockSizeRandKernel);

        // Dispatch CurandState
        // We need a dispatch that initializes hiprand per thread
        {
            dim3 gridSize(gridSizeRandKernel, 1, 1);
            dim3 blockSize(blockSizeRandKernel, 1, 1);
            size_t sharedMemorySizeBytes = 0;
            hipStream_t stream = 0;

            FullExperimentRunnerOptimalPerturbOptimized_GPU_InitializeCurandState << <gridSize, blockSize, sharedMemorySizeBytes, stream >> > (
                static_cast<uint32_t>(seed),
                m_pPerGlobalThreadRandStates,
                numGlobalPerturbThreads
            );

            CudaSafeErrorCheck(hipGetLastError(), "Rand state init kernal launch");
            CudaSafeErrorCheck(hipDeviceSynchronize(), "Rand state kernel sync");
        }


        std::cout << "\tUsed Device Memory Before: " << m_usedDeviceMemoryInBytes << std::endl;
        std::cout << "\tNewly allocated memory: " << usedMemoryInBytes << std::endl;

        m_usedDeviceMemoryInBytes += usedMemoryInBytes;

        std::cout << "\tUsed Device Memory After: " << m_usedDeviceMemoryInBytes << std::endl;

        return true;
    }
    
    void FullExperimentRunnerOptimalPerturbOptimized_GPU::CleanupCudaRandStates()
    {
        CudaSafeErrorCheck(hipFree((void*)m_pPerGlobalThreadRandStates),
            "RandState free");
    }


    // Pass in total number of threads that can be used, as well as the number of batches of 10^6 paths which will be generated
    bool FullExperimentRunnerOptimalPerturbOptimized_GPU::SetupCudaPerturb(uint32_t numGlobalPerturbThreads, uint32_t numCombinedWeightValues)
    {
        std::cout << "Setup Cuda Perturb: " << std::endl;
        uint64_t usedMemoryInBytes = 0;

        // Every global thread needs its own curve scratch space
        CudaSafeErrorCheck(hipMalloc((void**)&m_pPerGlobalThreadScratchSpacePositions, sizeof(float) * numGlobalPerturbThreads * m_upInitialCurve->m_numSegments * (3)),
            "Cuda malloc Scratch Space Positions");
        usedMemoryInBytes += (sizeof(float) * numGlobalPerturbThreads * m_upInitialCurve->m_numSegments * (3));

        // Every global thread needs its own curve scratch space left and right and working
        CudaSafeErrorCheck(hipMalloc((void**)&m_pPerGlobalThreadScratchSpaceTangents, sizeof(float) * numGlobalPerturbThreads * m_upInitialCurve->m_numSegments * (3)),
            "Cuda malloc Scratch Space Tangents");
        usedMemoryInBytes += (sizeof(float) * numGlobalPerturbThreads * m_upInitialCurve->m_numSegments * (3));

        // Every global thread needs its own curve scratch space left and right and working
        CudaSafeErrorCheck(hipMalloc((void**)&m_pPerGlobalThreadScratchSpaceCurvatures, sizeof(float) * numGlobalPerturbThreads * m_upInitialCurve->m_numSegments * (1)),
            "Cuda malloc Scratch Space Curvatures");
        usedMemoryInBytes += (sizeof(float) * numGlobalPerturbThreads * m_upInitialCurve->m_numSegments * (1));

        // CudaSafeErrorCheck(hipMalloc((void**)&m_pPerGlobalThreadWorkingScratchSpacePositions, sizeof(float) * numGlobalPerturbThreads * m_upInitialCurve->m_numSegments * (3)),
        //     "Cuda malloc  Working Scratch Space Positions");
        // usedMemoryInBytes += (sizeof(float) * numGlobalPerturbThreads * m_upInitialCurve->m_numSegments * (3));

        // CudaSafeErrorCheck(hipMalloc((void**)&m_pPerGlobalThreadWorkingScratchSpaceTangents, sizeof(float) * numGlobalPerturbThreads * m_upInitialCurve->m_numSegments * (3)),
        //     "Cuda malloc Working Scratch Space Tangents");
        // usedMemoryInBytes += (sizeof(float) * numGlobalPerturbThreads * m_upInitialCurve->m_numSegments * (3));

        // CudaSafeErrorCheck(hipMalloc((void**)&m_pPerGlobalThreadWorkingScratchSpaceCurvatures, sizeof(float) * numGlobalPerturbThreads * m_upInitialCurve->m_numSegments * (1)),
        //     "Cuda malloc Working Scratch Space Curvatures");
        // usedMemoryInBytes += (sizeof(float) * numGlobalPerturbThreads * m_upInitialCurve->m_numSegments * (1));

        CudaSafeErrorCheck(hipMalloc((void**)&m_pPerThreadCombinedWeightValues, sizeof(CombinedWeightValues_C) * numGlobalPerturbThreads),
            "Cuda malloc combined weight values per thread");
        usedMemoryInBytes += (sizeof(CombinedWeightValues_C) * numGlobalPerturbThreads);

        CudaSafeErrorCheck(hipMalloc((void**)&m_pFinalCombinedValues, sizeof(CombinedWeightValues_C) * numCombinedWeightValues),
            "Cuda malloc combined weight values per thread");
        usedMemoryInBytes += (sizeof(CombinedWeightValues_C) * numCombinedWeightValues);

        std::cout << "\tUsed Device Memory Before: " << m_usedDeviceMemoryInBytes << std::endl;
        std::cout << "\tNewly allocated memory: " << usedMemoryInBytes << std::endl;

        m_usedDeviceMemoryInBytes += usedMemoryInBytes;

        std::cout << "\tUsed Device Memory After: " << m_usedDeviceMemoryInBytes << std::endl;

        // Copy that data over to the gpu
        
        // Setup data structures
        twisty::PerturbUtils::BoundaryConditions boundaryConditions;
        boundaryConditions.arclength = m_upInitialCurve->m_arclength;
        boundaryConditions.m_startPos = m_upInitialCurve->m_basePos;
        boundaryConditions.m_startDir = m_upInitialCurve->m_baseTangent;
        boundaryConditions.m_endPos = m_upInitialCurve->m_targetPos;
        boundaryConditions.m_endDir = m_upInitialCurve->m_targetTangent;

        std::vector<Farlor::Vector3> initialCurvePositions(m_experimentParams.numSegmentsPerCurve + 1);
        std::vector<Farlor::Vector3> initialCurveTangents(m_experimentParams.numSegmentsPerCurve + 1);
        std::vector<float> initialCurveCurvatures(m_experimentParams.numSegmentsPerCurve);

        // Positions
        // Hard code the first two positions
        initialCurvePositions[0] = m_upInitialCurve->m_basePos;
        initialCurvePositions[1] = m_upInitialCurve->m_basePos + m_upInitialCurve->m_baseTangent.Normalized() * m_upInitialCurve->m_segmentLength;
        for (int64_t segmentIdx = 2; segmentIdx < m_experimentParams.numSegmentsPerCurve; ++segmentIdx)
        {
            initialCurvePositions[segmentIdx] = m_upInitialCurve->m_positions[segmentIdx];
        }
        // Hard code the final position
        initialCurvePositions[m_experimentParams.numSegmentsPerCurve] = m_upInitialCurve->m_targetPos;

        twisty::PerturbUtils::UpdateTangentsFromPos(initialCurvePositions.data(), initialCurveTangents.data(),
            m_upInitialCurve->m_numSegments, boundaryConditions);
        twisty::PerturbUtils::UpdateCurvaturesFromTangents(initialCurveTangents.data(), initialCurveCurvatures.data(),
            m_upInitialCurve->m_numSegments, boundaryConditions, m_experimentParams.weightingParameters);

        const int64_t NumPosPerCurve = (m_upInitialCurve->m_numSegments + 1);
        const int64_t NumTanPerCurve = (m_upInitialCurve->m_numSegments + 1);
        const int64_t NumCurvaturePerCurve = (m_upInitialCurve->m_numSegments);

        // TODO: Should this be intermixed somehow for better performance?
        uint64_t idx = 0;
        for (int64_t threadIdx = 0; threadIdx < numGlobalPerturbThreads; ++threadIdx)
        {
            for (int64_t posIdx = 0; posIdx < NumPosPerCurve; posIdx++)
            {
                hipMemcpy((void*)&m_pPerGlobalThreadScratchSpacePositions[idx], (void*)initialCurvePositions.data(), initialCurvePositions.size() * sizeof(float) * 3, hipMemcpyHostToDevice);
            }
            idx += initialCurvePositions.size() * 3;
        }

        for (int64_t threadIdx = 0; threadIdx < numGlobalPerturbThreads; ++threadIdx)
        {
            // Copy Tan
            for (int64_t tanIdx = 0; tanIdx < NumTanPerCurve; tanIdx++)
            {
                hipMemcpy((void*)&m_pPerGlobalThreadScratchSpaceTangents[idx], (void*)initialCurveTangents.data(), initialCurveTangents.size() * sizeof(float) * 3, hipMemcpyHostToDevice);
            }
            idx += initialCurveTangents.size() * 3;
        }

        for (int64_t threadIdx = 0; threadIdx < numGlobalPerturbThreads; ++threadIdx)
        {
            // Copy Curvatures
            for (int64_t curvatureIdx = 0; curvatureIdx < NumCurvaturePerCurve; curvatureIdx++)
            {
                hipMemcpy((void*)&m_pPerGlobalThreadScratchSpaceCurvatures[idx], (void*)initialCurveCurvatures.data(), initialCurveCurvatures.size() * sizeof(float) * 1, hipMemcpyHostToDevice);
            }

            idx += initialCurveCurvatures.size();
        }


        // TODO: Is this cache even used?
        std::vector<CombinedWeightValues_C> perThreadCombinedWeightValues(numGlobalPerturbThreads);
        hipMemcpy((void*)m_pPerThreadCombinedWeightValues, (void*)perThreadCombinedWeightValues.data(), perThreadCombinedWeightValues.size() * sizeof(CombinedWeightValues_C), hipMemcpyHostToDevice);

        std::vector<CombinedWeightValues_C> finalCombinedWeights(numCombinedWeightValues);
        hipMemcpy((void*)m_pFinalCombinedValues, (void*)finalCombinedWeights.data(), finalCombinedWeights.size() * sizeof(CombinedWeightValues_C), hipMemcpyHostToDevice);

        return true;
    }

    void FullExperimentRunnerOptimalPerturbOptimized_GPU::CleanupCudaPerturb()
    {
        CudaSafeErrorCheck(hipFree((void*)m_pFinalCombinedValues),
            "Cuda free combined weight values for final answer");

        CudaSafeErrorCheck(hipFree((void*)m_pPerThreadCombinedWeightValues),
            "Cuda free combined weight values per thread");

        // CudaSafeErrorCheck(hipFree((void*)m_pPerGlobalThreadWorkingScratchSpaceCurvatures),
        //     "Cuda free Working Scratch Space Curvatures");

        // CudaSafeErrorCheck(hipFree((void*)m_pPerGlobalThreadWorkingScratchSpaceTangents),
        //     "Cuda free Working Scratch Space Tangents");

        // CudaSafeErrorCheck(hipFree((void*)m_pPerGlobalThreadWorkingScratchSpacePositions),
        //     "Cuda free Working Scratch Space Positions");

        CudaSafeErrorCheck(hipFree((void*)m_pPerGlobalThreadScratchSpaceCurvatures),
            "Cuda free Left Scratch Space Curvatures");

        CudaSafeErrorCheck(hipFree((void*)m_pPerGlobalThreadScratchSpaceTangents),
            "Cuda free Left Scratch Space Tangents");

        CudaSafeErrorCheck(hipFree((void*)m_pPerGlobalThreadScratchSpacePositions),
            "Cuda free Left Scratch Space Positions");
    }



    __global__ void FullExperimentRunnerOptimalPerturbOptimized_GPU_InitializeCurandState(uint32_t seed, hiprandState_t* pStates, uint32_t maxNumStates)
    {
        // Unique index
        const uint32_t globalThreadIdx = threadIdx.x + blockIdx.x * blockDim.x;
        if (globalThreadIdx < maxNumStates)
        {
            hiprand_init(seed + globalThreadIdx, 0, 0, &pStates[globalThreadIdx]);
        }
    }

    __device__ double WeightCurveViaCurvatureLog10_CUDA(float* pCurvatureStart, uint32_t numCurvatures, double* pWeightIntegral, double ds,
        twisty::WeightingParameters weightingParams_cuda)
    {
        return 0.0;
    }

    __global__ void FullExperimentRunnerOptimalPerturbOptimized_GPU_GeometryRandomKernel(
        int64_t numCombinedWeightValuesTotal,
        int64_t numCombinedWeightValuesPerWarp,
        int64_t numPathsPerThread,
        int64_t numPathsToSkipPerThread,
        int64_t numSegmentsPerCurve,
        hiprandState_t* pCurandStates,
        float* pPerGlobalThreadScratchSpacePositions,
        float* pPerGlobalThreadScratchSpaceTangents,
        float* pPerGlobalThreadScratchSpaceCurvatures,
        CombinedWeightValues_C* pPerThreadCombinedWeightValues,
        CombinedWeightValues_C* pFinalCombinedValues,
        const twisty::WeightingParameters& weightingParams,
        const twisty::PerturbUtils::BoundaryConditions_CudaSafe& csBoundaryConditions,
        const double* pLookupTable
    )
    {
        const uint32_t globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;

        uint32_t numPathsAccepted = 0;

        const int32_t NumPosPerCurve = (numSegmentsPerCurve + 1);
        const int32_t NumTanPerCurve = (numSegmentsPerCurve + 1);
        const int32_t NumCurvaturesPerCurve = numSegmentsPerCurve;

        const int32_t CurrentThreadPosStartIdx = NumPosPerCurve * globalThreadIdx;
        const int32_t CurrentThreadTanStartIdx = NumTanPerCurve * globalThreadIdx;
        const int32_t CurrentThreadCurvatureStartIdx = NumCurvaturesPerCurve * globalThreadIdx;

        int64_t numToSkip = numPathsToSkipPerThread;

        // Ok, we want to loop over the outer batches first, the number per warp
        for (int64_t combinedWeightValuesWarpIdx = 0; combinedWeightValuesWarpIdx < numCombinedWeightValuesPerWarp; combinedWeightValuesWarpIdx++)
        {
            // We want to stop generating in this case
            if (combinedWeightValuesWarpIdx + numCombinedWeightValuesPerWarp * blockIdx.x >= numCombinedWeightValuesTotal)
            {
                return;
            }

            // We need a loop over the batches
            for (int64_t combinedWeightValuesThreadIdx = 0; combinedWeightValuesThreadIdx < numPathsPerThread; combinedWeightValuesThreadIdx++)
            {
                // Ok, now we first want to reset the combined weight stuff
                CombinedWeightValues_C_Reset(pPerThreadCombinedWeightValues[blockIdx.x * blockDim.x + threadIdx.x]);

                {

                    // This is the perturbation piece.
                    // Can we do this in place, most likely
                    // This will modify pCurrentThreadCurve
                    // Remember, the structure of this is:
                    // Pos_0, .,,, Pos_M, Pos_[M+1], Tan_0, ..., Tan_M
                    for (int64_t pathCount = 0; pathCount < (numToSkip + MaxNumPathsPerCombinedWeight); ++pathCount)
                    {
                        // Should be 2 - 180
                        int64_t diff = floorf(hiprand_uniform(&pCurandStates[globalThreadIdx]) * 178.0) + 2;

                        int64_t leftPointIndex = floorf(hiprand_uniform(&pCurandStates[globalThreadIdx]) * (numSegmentsPerCurve - 1 - diff - 1)) + 1;

                        int64_t rightPointIndex = leftPointIndex + diff;

                        assert((rightPointIndex - leftPointIndex) >= diff);
                        assert(leftPointIndex < rightPointIndex);

                        // We need two frames for each segment to get the new curvature and torsion.
                        // we need the frame left of the segment, as well as the frame right of the segment.
                        // The left point also will act as the origin for rotating the points between leftPoint and rightPoint
                        const float leftPoint_x = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx + leftPointIndex * 3 + 0];
                        const float leftPoint_y = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx + leftPointIndex * 3 + 1];
                        const float leftPoint_z = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx + leftPointIndex * 3 + 2];

                        const float rightPoint_x = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx + rightPointIndex * 3 + 0];
                        const float rightPoint_y = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx + rightPointIndex * 3 + 1];
                        const float rightPoint_z = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx + rightPointIndex * 3 + 2];

                        float N_x = (rightPoint_x - leftPoint_x);
                        float N_y = (rightPoint_y - leftPoint_y);
                        float N_z = (rightPoint_z - leftPoint_z);
                        float N_length = sqrt(N_x * N_x + N_y * N_y + N_z * N_z);
                        N_x /= N_length;
                        N_y /= N_length;
                        N_z /= N_length;

                        // Overwrite angle
                        double randRotationAngle = (hiprand_uniform(&pCurandStates[globalThreadIdx]) * 2.0 - 1.0) * TwistyPi;
                        float N[3] = { N_x, N_y, N_z };

                        // Rotation
                        {
                            float rotationMatrix[9] = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };
                            RotationMatrixAroundAxis(randRotationAngle, (float*)(N), rotationMatrix);

                            for (int64_t pointIdx = (leftPointIndex + 1); pointIdx < rightPointIndex; ++pointIdx)
                            {
                                float shiftedPoint[3];
                                shiftedPoint[0] = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx + pointIdx * 3 + 0] - leftPoint_x;
                                shiftedPoint[1] = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx + pointIdx * 3 + 1] - leftPoint_y;
                                shiftedPoint[2] = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx + pointIdx * 3 + 2] - leftPoint_z;

                                // Rotate and stuff back in shifted point
                                RotateVectorByMatrix(rotationMatrix, (float*)(shiftedPoint));
                                // Update the point with the rotated version
                                pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx + pointIdx * 3 + 0] = shiftedPoint[0] + leftPoint_x;
                                pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx + pointIdx * 3 + 1] = shiftedPoint[1] + leftPoint_y;
                                pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx + pointIdx * 3 + 2] = shiftedPoint[2] + leftPoint_z;
                            }

                            //Now, simply compute the difference in positions at the two edges of the rotated rigidbody.
                            //We can do a different approach later.
                            // Here, we want to do a perturb update call
                            twisty::PerturbUtils::UpdateTangentsFromPosCudaSafe(&pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx],
                                &pPerGlobalThreadScratchSpaceTangents[CurrentThreadTanStartIdx],
                                numSegmentsPerCurve, csBoundaryConditions);

                            twisty::PerturbUtils::UpdateCurvaturesFromTangentsCudaSafe(&pPerGlobalThreadScratchSpaceTangents[CurrentThreadTanStartIdx],
                                &pPerGlobalThreadScratchSpaceCurvatures[CurrentThreadCurvatureStartIdx],
                                numSegmentsPerCurve, csBoundaryConditions, weightingParams);
                        }

                        double pathWeightLog10 = 1.0; // = twisty::PathWeighting::WeightCurveViaCurvatureLog10_CUDA(&(pPerGlobalThreadLeftScratchSpaceCurvatures[CurrentThreadCurvatureStartIdx]),
                        //numSegmentsPerCurve, pLookupTable);

                        numPathsAccepted++;
                        if (pathCount < numPathsToSkipPerThread)
                        {
                            // Skip
                        }
                        else
                        {
                            // Else, contribute to the paths
                            CombinedWeightValues_C_AddValue(pPerThreadCombinedWeightValues[blockIdx.x * blockDim.x + threadIdx.x], pathWeightLog10);
                        }
                    }
                }
            }

            __syncthreads();

            if (threadIdx.x == 0)
            {
                CombinedWeightValues_C combinedResult;
                CombinedWeightValues_C_Reset(combinedResult);

                const uint32_t globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;

                for (uint32_t warpThreadIdx = 0; warpThreadIdx < blockDim.x; ++warpThreadIdx)
                {
                    combinedResult = CombinedWeightValues_C_CombineValues(combinedResult, pPerThreadCombinedWeightValues[blockIdx.x * blockDim.x + warpThreadIdx]);
                }

                // Finally, we write to the combined final values
                (pFinalCombinedValues[blockIdx.x * numCombinedWeightValuesPerWarp + combinedWeightValuesWarpIdx]) = combinedResult;//(pPerThreadCombinedWeightValues[blockIdx.x * blockDim.x + 0]);
            }

            __syncthreads();
        }
    }

}