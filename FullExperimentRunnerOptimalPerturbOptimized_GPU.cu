#include "hip/hip_runtime.h"
#include "FullExperimentRunnerOptimalPerturbOptimized_GPU.h"

#include <boost\multiprecision\cpp_dec_float.hpp>

#include "CombinedWeightUtils.h"
#include "CurvePerturbUtils.h"
#include "CurveUtils.h"
#include "MathConsts.h"
#include "PathWeightUtils.h"
#include "PathWeighters.h"

#include <hiprand.h>

#include <omp.h>

#include <assert.h>
#include <ctime>
#include <fstream>
#include <filesystem>
#include <limits>

#include <chrono>
#include <thread>
#include <mutex>
#include <condition_variable>
#include <atomic>
#include <stdlib.h>
#include <memory>

const uint32_t PerturbGridSize = 64;
const uint32_t PerturbBlockSize = 64;

namespace twisty {
static void CudaSafeErrorCheck(hipError_t error, std::string message)
{
    if (error != hipSuccess) {
        std::string errorString(hipGetErrorString(error));
        fprintf(stdout, "ERROR: %s : %s\n", message.c_str(), errorString.c_str());
        // assert(false);
    }
}
__global__ void FullExperimentRunnerOptimalPerturbOptimized_GPU_GeometryRandomKernel(
      const int64_t numCombinedWeightValuesTotal,
      const int64_t numCombinedWeightValuesPerWarp,
      const int64_t numPathsPerThread,
      const int64_t numPathsToSkipPerThread,
      const int64_t numSegmentsPerCurve,
      hiprandState_t *const pCurandStates,
      float *const pPerGlobalThreadScratchSpacePositions,
      float *const pPerGlobalThreadScratchSpaceTangents,
      float *const pPerGlobalThreadScratchSpaceCurvatures,
      CombinedWeightValues_C *const pFinalCombinedValues,
      const int32_t weightingMethod,
      const double pathNormalizerLog10,
      const twisty::PerturbUtils::BoundaryConditions_CudaSafe csBoundaryConditions,
      const double *const pWeightLookupTable,
      const int32_t weightLookupTableSize,
      const double ds,
      const double minCurvature,
      const double maxCurvature,
      const double curvatureStepSize);

FullExperimentRunnerOptimalPerturbOptimized_GPU::FullExperimentRunnerOptimalPerturbOptimized_GPU(
      ExperimentRunner::ExperimentParameters &experimentParams, Bootstrapper &bootstrapper)
    : ExperimentRunner(experimentParams, bootstrapper)
{
}

FullExperimentRunnerOptimalPerturbOptimized_GPU::~FullExperimentRunnerOptimalPerturbOptimized_GPU()
{
}

ExperimentRunner::RunnerSpecificResults
FullExperimentRunnerOptimalPerturbOptimized_GPU::RunnerSpecificRunExperiment()
{
    /* --------------------- */
    auto setupTimeStart = std::chrono::high_resolution_clock::now();
    /* --------------------- */

    assert(m_experimentParams.weightingParameters.scatterValues.size() > 0);

    // TODO: For now, we simply will support one scattering value
    if (m_experimentParams.weightingParameters.scatterValues.size() > 1) {
        std::cout << "WARNING: Only one scatter value supported, defaulting to first specified "
                     "scatter parameter"
                  << std::endl;
    }
    m_experimentParams.weightingParameters.scatter
          = m_experimentParams.weightingParameters.scatterValues[0];
    std::unique_ptr<twisty::PathWeighting::BaseWeightLookupTable> lookupEvaluator = nullptr;

    if (m_experimentParams.weightingParameters.weightingMethod
          == WeightingMethod::SimplifiedModel) {
        lookupEvaluator = std::make_unique<twisty::PathWeighting::SimpleWeightLookupTable>(
              m_experimentParams.weightingParameters, m_upInitialCurve->m_segmentLength);
    } else {
        lookupEvaluator = std::make_unique<twisty::PathWeighting::WeightLookupTableIntegral>(
              m_experimentParams.weightingParameters, m_upInitialCurve->m_segmentLength);
    }

    lookupEvaluator->ExportValues(m_experimentDirPath.string());

    twisty::PerturbUtils::BoundaryConditions boundaryConditions
          = m_upInitialCurve->GetBoundaryConditions();

    bool result = SetupCudaDevice();
    if (!result) {
        printf("Failed to setup cuda device\n");
        return {};
    }

    // Calculate number of paths needed to generate

    const uint32_t warpPathCount = MaxNumPathsPerCombinedWeight;
    const uint32_t numGlobalPerturbThreads = PerturbGridSize * PerturbBlockSize;

    const uint32_t numCombinedWeightValuesTotal
          = (m_experimentParams.numPathsInExperiment + MaxNumPathsPerCombinedWeight - 1)
          / MaxNumPathsPerCombinedWeight;
    const uint32_t numCombinedWeightValuesPerWarp
          = (numCombinedWeightValuesTotal + PerturbGridSize - 1) / PerturbGridSize;
    const uint32_t numPathsPerThread
          = (MaxNumPathsPerCombinedWeight + PerturbBlockSize - 1) / PerturbBlockSize;

    std::cout << "Num Global Perturb Threads: " << numGlobalPerturbThreads << std::endl;
    std::cout << "numPathsInExperiment: " << m_experimentParams.numPathsInExperiment << std::endl;
    std::cout << "numPathsPerBatch: " << warpPathCount << std::endl;
    std::cout << "Num Combined Weight Values Total: " << numCombinedWeightValuesTotal << std::endl;
    std::cout << "Num Combined weights per warp: " << numCombinedWeightValuesPerWarp << std::endl;
    std::cout << "Perturb Grid Size required: " << PerturbGridSize << std::endl;
    std::cout << "Perturb Block Size required: " << PerturbBlockSize << std::endl;

    int32_t seed = m_experimentParams.curvePurturbSeed;
    if (seed == 0) {
        seed = time(0);
    }

    auto setupCuRandTimeStart = std::chrono::high_resolution_clock::now();
    {
        result = SetupCuRandStates(numGlobalPerturbThreads, seed);
        if (!result) {
            printf("Failed to setup CuRand states\n");
            return {};
        }
    }
    auto setupCuRandTimeEnd = std::chrono::high_resolution_clock::now();

    auto setupCudaPerturbStart = std::chrono::high_resolution_clock::now();
    {
        result = SetupCudaPerturb(numGlobalPerturbThreads, numCombinedWeightValuesTotal,
              m_experimentParams.numSegmentsPerCurve, lookupEvaluator->AccessLookupTable());
        if (!result) {
            printf("Failed to setup Cuda Perturb\n");
            return {};
        }
    }
    auto setupCudaPerturbEnd = std::chrono::high_resolution_clock::now();

    auto experimentTimeStart = std::chrono::high_resolution_clock::now();

    std::stringstream fnFilenameSS;
    fnFilenameSS << "SavedFN";
    fnFilenameSS << m_experimentParams.numSegmentsPerCurve;
    fnFilenameSS << ".fnd";
    const std::filesystem::path fnFilePath = std::filesystem::current_path() / fnFilenameSS.str();
    std::unique_ptr<PathWeighting::NormalizerStuff::BaseNormalizer> upFN = nullptr;

    // We dont need this actually, so we can just load the default one
    //     {
    //         // If we can load the fn data, load it
    //         if (std::filesystem::exists(fnFilePath)) {
    //             std::cout << "Using cached fd file at: " << fnFilePath << std::endl;
    //             std::ifstream inFile(fnFilePath);
    //             upFN = std::make_unique<PathWeighting::NormalizerStuff::FN>(inFile);
    //             inFile.close();
    //         }
    //         // We need to generate it this time and save it off to use next time
    //         else {
    //             // This is the max M value
    //             const int maxorder = m_upInitialCurve->m_numSegments;

    //             // Generate the fn data table
    //             const int numZSamples = 5000;
    //             const int numIntegrationSamples = 5000;

    //             // Arbitrarily set min and max |r_vec| values.
    //             // Why this specific max bound, I do not know
    //             const double rMin = 0.0;
    //             const double rMax = 200.0;
    //             upFN = std::make_unique<PathWeighting::NormalizerStuff::FN>(
    //                   numZSamples, numIntegrationSamples, maxorder, rMin, rMax);

    //             std::ofstream outFile(fnFilePath);
    //             dynamic_cast<PathWeighting::NormalizerStuff::FN *>(upFN.get())->WriteToFile(outFile);
    //             outFile.close();
    //         }
    //     }
    //     PathWeighting::NormalizerStuff::BaseNormalizer &fn = (*upFN);

    // Why the 1/(delta s) = (M+2)/s?
    //     Farlor::Vector3 Z = (boundaryConditions.m_endPos - boundaryConditions.m_startPos)
    //                 * (m_upInitialCurve->m_numSegments + 2) / boundaryConditions.arclength
    //           - boundaryConditions.m_endDir - boundaryConditions.m_startDir;
    //     std::cout << "Z: " << Z << std::endl;
    //     std::cout << "|Z|: " << Z.Magnitude() << std::endl;

    PathWeighting::NormalizerStuff::NormalizerDoubleType pathNormalizer = 1.0;
    if (m_experimentParams.weightingParameters.weightingMethod
          == WeightingMethod::RadiativeTransfer) {
        pathNormalizer = PathWeighting::NormalizerStuff::Norm(m_upInitialCurve->m_numSegments,
              m_upInitialCurve->m_segmentLength, boundaryConditions);
    }
    const boost::multiprecision::cpp_dec_float_100 pathNormalizerLog10
          = boost::multiprecision::log10(pathNormalizer);

    std::cout << "PathNormalizer: " << pathNormalizer << std::endl;
    std::cout << "PathNormalizerLog10: " << pathNormalizerLog10 << std::endl;

    auto setupTimeEnd = std::chrono::high_resolution_clock::now();
    /* --------------------- */


    /* --------------------- */

    uint64_t perturbTimeCount = 0;
    uint64_t weightCalcTimeCount = 0;

    std::cout << "numPathsInExperiment specified: " << m_experimentParams.numPathsInExperiment
              << std::endl;

    std::cout << "numPathsInExperiment generated: "
              << numCombinedWeightValuesTotal * MaxNumPathsPerCombinedWeight << std::endl;
    std::cout << "numCombinedWeightValuesTotal: " << numCombinedWeightValuesTotal << std::endl;
    std::cout << "numCombinedWeightValuesPerWarp: " << numCombinedWeightValuesPerWarp << std::endl;
    std::cout << "numPathsPerThread: " << numPathsPerThread << std::endl;

    auto perturbTimeStart = std::chrono::high_resolution_clock::now();

    twisty::PerturbUtils::BoundaryConditions_CudaSafe csBoundaryConditions;
    csBoundaryConditions.m_startPos[0] = boundaryConditions.m_startPos[0];
    csBoundaryConditions.m_startPos[1] = boundaryConditions.m_startPos[1];
    csBoundaryConditions.m_startPos[2] = boundaryConditions.m_startPos[2];

    csBoundaryConditions.m_startDir[0] = boundaryConditions.m_startDir[0];
    csBoundaryConditions.m_startDir[1] = boundaryConditions.m_startDir[1];
    csBoundaryConditions.m_startDir[2] = boundaryConditions.m_startDir[2];

    csBoundaryConditions.m_endPos[0] = boundaryConditions.m_endPos[0];
    csBoundaryConditions.m_endPos[1] = boundaryConditions.m_endPos[1];
    csBoundaryConditions.m_endPos[2] = boundaryConditions.m_endPos[2];

    csBoundaryConditions.m_endDir[0] = boundaryConditions.m_endDir[0];
    csBoundaryConditions.m_endDir[1] = boundaryConditions.m_endDir[1];
    csBoundaryConditions.m_endDir[2] = boundaryConditions.m_endDir[2];

    csBoundaryConditions.arclength = boundaryConditions.arclength;

    {
        dim3 gridSize(PerturbGridSize, 1, 1);
        dim3 blockSize(PerturbBlockSize, 1, 1);
        size_t sharedMemorySizeBytes = 0;
        hipStream_t stream = 0;

        std::cout << "Dispatching with: " << std::endl;
        std::cout << "\tGrid Size: " << PerturbGridSize << std::endl;
        std::cout << "\tBlock Size: " << PerturbBlockSize << std::endl;


        std::cout << "Weight Table Ptr: " << lookupEvaluator->AccessLookupTable().data()
                  << std::endl;
        std::cout << "Weight Table Size: " << lookupEvaluator->AccessLookupTable().size()
                  << std::endl;
        std::cout << "DS: " << lookupEvaluator->GetDs() << std::endl;
        std::cout << "Min Curvature: " << lookupEvaluator->GetMinCurvature() << std::endl;
        std::cout << "Max Curvature: " << lookupEvaluator->GetMaxCurvature() << std::endl;
        std::cout << "Curvature Step Size: " << lookupEvaluator->GetCurvatureStepSize()
                  << std::endl;

        FullExperimentRunnerOptimalPerturbOptimized_GPU_GeometryRandomKernel<<<gridSize, blockSize,
              sharedMemorySizeBytes, stream>>>(numCombinedWeightValuesTotal,
              numCombinedWeightValuesPerWarp,
              numPathsPerThread,
              m_experimentParams.numPathsToSkip,
              m_experimentParams.numSegmentsPerCurve,
              m_pPerGlobalThreadRandStates,
              m_pPerGlobalThreadScratchSpacePositions,
              m_pPerGlobalThreadScratchSpaceTangents,
              m_pPerGlobalThreadScratchSpaceCurvatures,
              m_pFinalCombinedValues,
              (int32_t)m_experimentParams.weightingParameters.weightingMethod,
              m_experimentParams.weightingParameters.weightingMethod
                          == twisty::WeightingMethod::RadiativeTransfer
                    ? pathNormalizerLog10.convert_to<double>()
                    : 0.0,
              csBoundaryConditions,
              m_pDeviceWeightLookupTable,
              lookupEvaluator->AccessLookupTable().size(),
              lookupEvaluator->GetDs(),
              lookupEvaluator->GetMinCurvature(),
              lookupEvaluator->GetMaxCurvature(),
              lookupEvaluator->GetCurvatureStepSize());

        CudaSafeErrorCheck(hipGetLastError(), "GPU_GeometryRandomKernel kernal launch");
        CudaSafeErrorCheck(hipDeviceSynchronize(), "GPU_GeometryRandomKernel kernel sync");
    }

    auto perturbTimeEnd = std::chrono::high_resolution_clock::now();
    perturbTimeCount
          = std::chrono::duration_cast<std::chrono::milliseconds>(perturbTimeEnd - perturbTimeStart)
                  .count();

    // -------------------
    auto weightingTimeStart = std::chrono::high_resolution_clock::now();

    printf("Copying values back\n");
    // Copying back values
    std::vector<CombinedWeightValues_C> combinedWeightValues(numCombinedWeightValuesTotal);
    CudaSafeErrorCheck(hipMemcpy(combinedWeightValues.data(), m_pFinalCombinedValues,
                             sizeof(CombinedWeightValues_C) * numCombinedWeightValuesTotal,
                             hipMemcpyDeviceToHost),
          "Copy combined values back from GPU");

    printf("Done Copying values back\n");

    // We need to calculate the absorbtion/scattering piece
    boost::multiprecision::cpp_dec_float_100 bigTotalExperimentWeight = 0.0;

    uint64_t numWeightsGenerated = 0;

    // No, we calculating the weighting
    for (auto &combinedWeightValue : combinedWeightValues) {
        // std::cout << "Combined Weight Value" << std::endl;
        // std::cout << "Extracted Value: " << ExtractFinalValue(combinedWeightValue);
        // std::cout << "\tNum Values: " << combinedWeightValue.m_numValues << std::endl;
        // std::cout << "\tOffset: " << combinedWeightValue.m_offset << std::endl;
        // std::cout << "\tRunning Total: " << combinedWeightValue.m_runningTotal << std::endl;

        const boost::multiprecision::cpp_dec_float_100 extractedDispatchWeight
              = ExtractFinalValue(combinedWeightValue);
        bigTotalExperimentWeight += extractedDispatchWeight;

        if (m_experimentParams.outputBigFloatWeights) {
            UpdateConvergenceWeight(combinedWeightValue.m_numValues, extractedDispatchWeight);
        }

        numWeightsGenerated += combinedWeightValue.m_numValues;
    }
    // bigTotalExperimentWeight *= pathNormalizer;

    auto weightingTimeEnd = std::chrono::high_resolution_clock::now();
    weightCalcTimeCount = std::chrono::duration_cast<std::chrono::milliseconds>(
          weightingTimeEnd - weightingTimeStart)
                                .count();
    /* --------------------- */

    // Cleanup stuff

    {
        CleanupCudaPerturb();
        CleanupCudaRandStates();
        CleanupCudaDevice();
    }

    auto timeMs
          = std::chrono::duration_cast<std::chrono::milliseconds>(setupTimeEnd - setupTimeStart);

    ExperimentResults results;
    results.experimentWeights.push_back(bigTotalExperimentWeight);
    results.totalPathsGenerated = numWeightsGenerated;
    results.numFailedPaths = 0;

    ExperimentRunner::RunnerSpecificResults specificResult;
    specificResult.experimentResults = std::make_optional<ExperimentResults>(results);
    specificResult.setupMs = timeMs.count();
    specificResult.runExperimentMs = perturbTimeCount;
    specificResult.weightingMs = weightCalcTimeCount;

    return specificResult;
}

bool FullExperimentRunnerOptimalPerturbOptimized_GPU::SetupCudaDevice()
{
    // Query for the number of devices avalible
    int32_t numDevices = 0;
    CudaSafeErrorCheck(hipGetDeviceCount(&numDevices), "Get Devices");

    // We need at least one cuda device
    if (numDevices <= 0) {
        printf("No CUDA device avalible. Cannot execute experiment.\n");
        return false;
    }

    // Iterate over all devices and report the device stats.
    for (int32_t i = 0; i < numDevices; ++i) {
        hipDeviceProp_t prop;
        CudaSafeErrorCheck(hipGetDeviceProperties(&prop, i), "Get Device Prop");
        printf("\nDevice Number: %d\n", i);
        printf("\tDevice name: %s\n", prop.name);
        printf("\tSM Count: %d\n", prop.multiProcessorCount);
        printf("\tSM Shared Memory: %d\n", prop.sharedMemPerBlock);
        printf("\tWarp Size: %d\n", prop.warpSize);
        printf("\tThreads per SM: %d\n", prop.maxThreadsPerMultiProcessor);
        printf("\tPeak Memory Bandwidth (GB/s): %f\n",
              2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
        printf("\tGlobal Memory: %zu\n", prop.totalGlobalMem);
        printf("\tConst Memory: %zu\n", prop.totalConstMem);
    }

    // We select the first device only
    const uint32_t selectedDeviceIdx = 0;
    std::cout << "\nSelected device: " << selectedDeviceIdx << std::endl;
    hipDeviceProp_t deviceProp;
    CudaSafeErrorCheck(hipGetDeviceProperties(&deviceProp, 0), "Get first device prop");

    m_numSMs = deviceProp.multiProcessorCount;
    m_warpSize = deviceProp.warpSize;
    m_maxThreadsPerMultiprocessor = deviceProp.maxThreadsPerMultiProcessor;

    return true;
}

void FullExperimentRunnerOptimalPerturbOptimized_GPU::CleanupCudaDevice() { }

bool FullExperimentRunnerOptimalPerturbOptimized_GPU::SetupCuRandStates(
      int32_t numGlobalPerturbThreads, uint32_t seed)
{
    std::cout << "Setup Cuda Perturb: " << std::endl;
    uint64_t usedMemoryInBytes = 0;

    // Random Seed Kernel
    // Every block thread needs its own hiprand state
    CudaSafeErrorCheck(hipMalloc((void **)&m_pPerGlobalThreadRandStates,
                             sizeof(hiprandState_t) * numGlobalPerturbThreads),
          "RandState malloc");
    usedMemoryInBytes += (sizeof(hiprandState_t) * numGlobalPerturbThreads);

    int blockSizeRandKernel = 0;
    int minGridSizeRandKernel = 0;
    size_t sharedMemoryUse = 0;
    size_t maxBlockSize = 0;
    CudaSafeErrorCheck(
          hipOccupancyMaxPotentialBlockSize(&minGridSizeRandKernel, &blockSizeRandKernel,
                FullExperimentRunnerOptimalPerturbOptimized_GPU_InitializeCurandState,
                sharedMemoryUse, maxBlockSize),
          "Failed to calculating occupancy for InitializeCuRandState kernel");
    std::cout << "\nInitializeCurandState: " << std::endl;
    std::cout << "\tBlock Size: " << blockSizeRandKernel << std::endl;
    std::cout << "\tMin Grid Size: " << minGridSizeRandKernel << std::endl;

    size_t gridSizeRandKernel
          = ((numGlobalPerturbThreads) + blockSizeRandKernel - 1) / blockSizeRandKernel;

    printf("\tInitializeCurandState Grid Size: %d\n", gridSizeRandKernel);
    printf("\tInitializeCurandState Block Size: %d\n", blockSizeRandKernel);

    // Dispatch CurandState
    // We need a dispatch that initializes hiprand per thread
    {
        dim3 gridSize(gridSizeRandKernel, 1, 1);
        dim3 blockSize(blockSizeRandKernel, 1, 1);
        size_t sharedMemorySizeBytes = 0;
        hipStream_t stream = 0;

        FullExperimentRunnerOptimalPerturbOptimized_GPU_InitializeCurandState<<<gridSize, blockSize,
              sharedMemorySizeBytes, stream>>>(
              static_cast<uint32_t>(seed), m_pPerGlobalThreadRandStates, numGlobalPerturbThreads);

        CudaSafeErrorCheck(hipGetLastError(), "Rand state init kernal launch");
        CudaSafeErrorCheck(hipDeviceSynchronize(), "Rand state kernel sync");
    }


    std::cout << "\tUsed Device Memory Before: " << m_usedDeviceMemoryInBytes << std::endl;
    std::cout << "\tNewly allocated memory: " << usedMemoryInBytes << std::endl;

    m_usedDeviceMemoryInBytes += usedMemoryInBytes;

    std::cout << "\tUsed Device Memory After: " << m_usedDeviceMemoryInBytes << std::endl;

    return true;
}

void FullExperimentRunnerOptimalPerturbOptimized_GPU::CleanupCudaRandStates()
{
    CudaSafeErrorCheck(hipFree((void *)m_pPerGlobalThreadRandStates), "RandState free");
}


// Pass in total number of threads that can be used, as well as the number of batches of 10^6 paths which will be generated
bool FullExperimentRunnerOptimalPerturbOptimized_GPU::SetupCudaPerturb(
      int32_t numGlobalPerturbThreads, int32_t numCombinedWeightValues, int32_t numSegments,
      const std::vector<double> &weightTable)
{
    std::cout << "Setup Cuda Perturb: " << std::endl;
    uint64_t usedMemoryInBytes = 0;

    // Every global thread needs its own curve scratch space
    const uint64_t positionBytes = numGlobalPerturbThreads * (numSegments + 1) * sizeof(float) * 3;
    CudaSafeErrorCheck(hipMalloc((void **)&m_pPerGlobalThreadScratchSpacePositions, positionBytes),
          "Cuda malloc Scratch Space Positions");
    usedMemoryInBytes += positionBytes;

    // Every global thread needs its own curve scratch space left and right and working
    const uint64_t tangentBytes = numGlobalPerturbThreads * numSegments * sizeof(float) * 3;
    CudaSafeErrorCheck(hipMalloc((void **)&m_pPerGlobalThreadScratchSpaceTangents, tangentBytes),
          "Cuda malloc Scratch Space Tangents");
    usedMemoryInBytes += tangentBytes;

    // Every global thread needs its own curve scratch space left and right and working
    const uint64_t curvatureBytes = numGlobalPerturbThreads * (numSegments - 1) * sizeof(float);
    CudaSafeErrorCheck(
          hipMalloc((void **)&m_pPerGlobalThreadScratchSpaceCurvatures, curvatureBytes),
          "Cuda malloc Scratch Space Curvatures");
    usedMemoryInBytes += curvatureBytes;

    const uint64_t combinedValueBytes = sizeof(CombinedWeightValues_C) * numCombinedWeightValues;
    CudaSafeErrorCheck(hipMalloc((void **)&m_pFinalCombinedValues, combinedValueBytes),
          "Cuda malloc combined weight values per thread");
    usedMemoryInBytes += combinedValueBytes;

    const uint64_t weightTableBytes = sizeof(double) * weightTable.size();
    CudaSafeErrorCheck(hipMalloc((void **)&m_pDeviceWeightLookupTable, weightTableBytes),
          "Cuda malloc combined weight values per thread");
    usedMemoryInBytes += weightTableBytes;

    std::cout << "\tUsed Device Memory Before: " << m_usedDeviceMemoryInBytes << std::endl;
    std::cout << "\tNewly allocated memory: " << usedMemoryInBytes << std::endl;

    m_usedDeviceMemoryInBytes += usedMemoryInBytes;

    std::cout << "\tUsed Device Memory After: " << m_usedDeviceMemoryInBytes << std::endl;

    // Copy that data over to the gpu

    // Setup data structures
    twisty::PerturbUtils::BoundaryConditions_CudaSafe boundaryConditionsCudaSafe
          = m_upInitialCurve->GetBoundaryConditionsCudaSafe();

    // Setup data structures
    float *pInitialCurvePositions = nullptr;
    float *pInitialCurveTangents = nullptr;
    float *pInitialCurveCurvatures = nullptr;
    hipHostMalloc(&pInitialCurvePositions, (numSegments + 1) * sizeof(float) * 3);
    hipHostMalloc(&pInitialCurveTangents, numSegments * sizeof(float) * 3);
    hipHostMalloc(&pInitialCurveCurvatures, (numSegments - 1) * sizeof(float));

    memcpy(pInitialCurvePositions, (float *)m_upInitialCurve->m_positions.data(),
          (numSegments + 1) * sizeof(float) * 3);

    // Update and curvature
    twisty::PerturbUtils::UpdateTangentsFromPosCudaSafe(pInitialCurvePositions,
          pInitialCurveTangents, m_upInitialCurve->m_numSegments, boundaryConditionsCudaSafe);
    twisty::PerturbUtils::UpdateCurvaturesFromTangentsCudaSafe(pInitialCurveTangents,
          pInitialCurveCurvatures, m_upInitialCurve->m_numSegments, boundaryConditionsCudaSafe,
          (int32_t)m_experimentParams.weightingParameters.weightingMethod);

    // TODO: Should this be intermixed somehow for better performance?
    std::cout << "Copying over intial curves" << std::endl;
    std::cout << "\tNum Global Perturb Threads: " << numGlobalPerturbThreads << std::endl;

    {
        uint64_t idx = 0;
        for (int64_t threadIdx = 0; threadIdx < numGlobalPerturbThreads; ++threadIdx) {
            CudaSafeErrorCheck(hipMemcpy((void *)&(m_pPerGlobalThreadScratchSpacePositions[idx]),
                                     (void *)pInitialCurvePositions,
                                     (numSegments + 1) * sizeof(float) * 3,
                                     hipMemcpyHostToDevice),
                  "Copy inital positions to per thread scratch space");
            idx += (numSegments + 1) * 3;
        }
    }

    {
        uint64_t idx = 0;
        for (int64_t threadIdx = 0; threadIdx < numGlobalPerturbThreads; ++threadIdx) {
            CudaSafeErrorCheck(hipMemcpy((void *)&(m_pPerGlobalThreadScratchSpaceTangents[idx]),
                                     (void *)pInitialCurveTangents,
                                     numSegments * sizeof(float) * 3,
                                     hipMemcpyHostToDevice),
                  "Copy inital tangents to per thread scratch space");
            idx += numSegments * 3;
        }
    }

    {
        uint64_t idx = 0;
        for (int64_t threadIdx = 0; threadIdx < numGlobalPerturbThreads; ++threadIdx) {
            CudaSafeErrorCheck(hipMemcpy((void *)&(m_pPerGlobalThreadScratchSpaceCurvatures[idx]),
                                     (void *)pInitialCurveCurvatures,
                                     (numSegments - 1) * sizeof(float),
                                     hipMemcpyHostToDevice),
                  "Copy inital curvatures to per thread scratch space");
            idx += (numSegments - 1);
        }
    }

    std::vector<CombinedWeightValues_C> finalCombinedWeights(numCombinedWeightValues);
    for (int i = 0; i < finalCombinedWeights.size(); i++) {
        CombinedWeightValues_C_Reset(finalCombinedWeights[i]);
    }
    hipMemcpy((void *)m_pFinalCombinedValues, (void *)finalCombinedWeights.data(),
          finalCombinedWeights.size() * sizeof(CombinedWeightValues_C), hipMemcpyHostToDevice);

    hipMemcpy((void *)m_pDeviceWeightLookupTable, (void *)weightTable.data(),
          weightTable.size() * sizeof(double), hipMemcpyHostToDevice);

    return true;
}

void FullExperimentRunnerOptimalPerturbOptimized_GPU::CleanupCudaPerturb()
{
    CudaSafeErrorCheck(hipFree((void *)m_pFinalCombinedValues),
          "Cuda free combined weight values for final answer");

    CudaSafeErrorCheck(hipFree((void *)m_pPerGlobalThreadScratchSpaceCurvatures),
          "Cuda free Left Scratch Space Curvatures");

    CudaSafeErrorCheck(hipFree((void *)m_pPerGlobalThreadScratchSpaceTangents),
          "Cuda free Left Scratch Space Tangents");

    CudaSafeErrorCheck(hipFree((void *)m_pPerGlobalThreadScratchSpacePositions),
          "Cuda free Left Scratch Space Positions");
}

__global__ void FullExperimentRunnerOptimalPerturbOptimized_GPU_InitializeCurandState(
      uint32_t seed, hiprandState_t *pStates, uint32_t maxNumStates)
{
    // Unique index
    const uint32_t globalThreadIdx = threadIdx.x + blockIdx.x * blockDim.x;
    if (globalThreadIdx < maxNumStates) {
        hiprand_init(seed + globalThreadIdx, 0, 0, &pStates[globalThreadIdx]);
    }
}

__device__ double WeightCurveViaCurvatureLog10_CUDA(float *pCurvatureStart, uint32_t numCurvatures,
      double *pWeightIntegral, double ds, twisty::WeightingParameters weightingParams_cuda)
{
    return 0.0;
}

__global__ void FullExperimentRunnerOptimalPerturbOptimized_GPU_GeometryRandomKernel(
      const int64_t numCombinedWeightValuesTotal,
      const int64_t numCombinedWeightValuesPerWarp,
      const int64_t numPathsPerThread,
      const int64_t numPathsToSkipPerThread,
      const int64_t numSegmentsPerCurve,
      hiprandState_t *const pCurandStates,
      float *const pPerGlobalThreadScratchSpacePositions,
      float *const pPerGlobalThreadScratchSpaceTangents,
      float *const pPerGlobalThreadScratchSpaceCurvatures,
      CombinedWeightValues_C *const pFinalCombinedValues,
      const int32_t weightingMethod,
      const double pathNormalizerLog10,
      const twisty::PerturbUtils::BoundaryConditions_CudaSafe csBoundaryConditions,
      const double *const pWeightLookupTable,
      const int32_t weightLookupTableSize,
      const double ds,
      const double minCurvature,
      const double maxCurvature,
      const double curvatureStepSize)
{
    __shared__ CombinedWeightValues_C perThreadWeightValues[PerturbBlockSize];

    // Should be between 0 and max num threads - 1
    volatile uint32_t globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;

    volatile int32_t NumPosPerCurve = (numSegmentsPerCurve + 1);
    volatile int32_t NumTanPerCurve = numSegmentsPerCurve;
    volatile int32_t NumCurvaturesPerCurve = (numSegmentsPerCurve - 1);

    volatile int32_t CurrentThreadPosStartIdx = NumPosPerCurve * 3 * globalThreadIdx;
    volatile int32_t CurrentThreadTanStartIdx = NumTanPerCurve * 3 * globalThreadIdx;
    volatile int32_t CurrentThreadCurvatureStartIdx = NumCurvaturesPerCurve * globalThreadIdx;

    // Ok, we want to loop over the outer batches first, the number per warp
    for (int64_t combinedWeightValuesWarpIdx = 0;
          combinedWeightValuesWarpIdx < numCombinedWeightValuesPerWarp;
          combinedWeightValuesWarpIdx++) {
        CombinedWeightValues_C_Reset(perThreadWeightValues[threadIdx.x]);

        // We want to stop generating in this case
        if (combinedWeightValuesWarpIdx + numCombinedWeightValuesPerWarp * blockIdx.x
              >= numCombinedWeightValuesTotal) {
            // printf("Exiting early\n");
            continue;
        }

        // We need a loop over the batches
        for (int64_t pathCount = 0; pathCount < numPathsToSkipPerThread + numPathsPerThread;
              pathCount++) {
            int64_t currentPathIdx
                  = numPathsPerThread * threadIdx.x + pathCount - numPathsToSkipPerThread;

            // Do our random rolls here
            float e0 = hiprand_uniform(&(pCurandStates[globalThreadIdx]));
            float e1 = hiprand_uniform(&(pCurandStates[globalThreadIdx]));
            float e2 = hiprand_uniform(&(pCurandStates[globalThreadIdx]));

            // We can exit once this point is reached as we have generated all the paths necessary for this thread
            if (currentPathIdx >= MaxNumPathsPerCombinedWeight) {
                printf("Breaking early as paths are done for this combined weight value\n");
                // We dont want to continue if we have already generated the correct number of paths.
                break;
            }

            // Ok, now we first want to reset the combined weight stuff
            {
                // This is the perturbation piece.
                // Can we do this in place, most likely
                // This will modify pCurrentThreadCurve
                // Remember, the structure of this is:
                // Pos_0, .,,, Pos_M, Pos_[M+1], Tan_0, ..., Tan_M
                {
                    // Should be 2 - 180
                    int64_t maxDiff = min((int)(numSegmentsPerCurve - 2), 25);
                    int64_t diff = floorf(e0 * (maxDiff - 2)) + 2;

                    // -2 from the -1 to offset for the +1, and -1 as required by index
                    int64_t leftPointIndex = floorf(e1 * (numSegmentsPerCurve - diff - 2)) + 1;

                    int64_t rightPointIndex = leftPointIndex + diff;

                    // We need two frames for each segment to get the new curvature and torsion.
                    // we need the frame left of the segment, as well as the frame right of the segment.
                    // The left point also will act as the origin for rotating the points between leftPoint and rightPoint
                    const float leftPoint_x
                          = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx
                                + leftPointIndex * 3 + 0];
                    const float leftPoint_y
                          = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx
                                + leftPointIndex * 3 + 1];
                    const float leftPoint_z
                          = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx
                                + leftPointIndex * 3 + 2];

                    const float rightPoint_x
                          = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx
                                + rightPointIndex * 3 + 0];
                    const float rightPoint_y
                          = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx
                                + rightPointIndex * 3 + 1];
                    const float rightPoint_z
                          = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx
                                + rightPointIndex * 3 + 2];


                    float N[3] = { rightPoint_x - leftPoint_x, rightPoint_y - leftPoint_y,
                        rightPoint_z - leftPoint_z };
                    volatile float N_length = sqrt(N[0] * N[0] + N[1] * N[1] + N[2] * N[2]);
                    N[0] /= N_length;
                    N[1] /= N_length;
                    N[2] /= N_length;

                    // Overwrite angle
                    const float randRotationAngle = (e2 * 2.0 - 1.0) * TwistyPi;

                    // Rotation
                    {
                        float rotationMatrix[9]
                              = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };
                        RotationMatrixAroundAxis(
                              randRotationAngle, (float *)N, (float *)rotationMatrix);

                        for (int64_t pointIdx = (leftPointIndex + 1); pointIdx < rightPointIndex;
                              ++pointIdx) {
                            float shiftedPoint[3];
                            shiftedPoint[0]
                                  = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx
                                          + pointIdx * 3 + 0]
                                  - leftPoint_x;
                            shiftedPoint[1]
                                  = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx
                                          + pointIdx * 3 + 1]
                                  - leftPoint_y;
                            shiftedPoint[2]
                                  = pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx
                                          + pointIdx * 3 + 2]
                                  - leftPoint_z;

                            // Rotate and stuff back in shifted point
                            RotateVectorByMatrix((float *)rotationMatrix, (float *)shiftedPoint);
                            // Update the point with the rotated version
                            pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx
                                  + pointIdx * 3 + 0]
                                  = shiftedPoint[0] + leftPoint_x;
                            pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx
                                  + pointIdx * 3 + 1]
                                  = shiftedPoint[1] + leftPoint_y;
                            pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx
                                  + pointIdx * 3 + 2]
                                  = shiftedPoint[2] + leftPoint_z;
                        }

                        // //Now, simply compute the difference in positions at the two edges of the rotated rigidbody.
                        // //We can do a different approach later.
                        // // Here, we want to do a perturb update call
                        twisty::PerturbUtils::UpdateTangentsFromPosCudaSafe(
                              &(pPerGlobalThreadScratchSpacePositions[CurrentThreadPosStartIdx]),
                              &(pPerGlobalThreadScratchSpaceTangents[CurrentThreadTanStartIdx]),
                              numSegmentsPerCurve, csBoundaryConditions);

                        twisty::PerturbUtils::UpdateCurvaturesFromTangentsCudaSafe(
                              &(pPerGlobalThreadScratchSpaceTangents[CurrentThreadTanStartIdx]),
                              &(pPerGlobalThreadScratchSpaceCurvatures
                                          [CurrentThreadCurvatureStartIdx]),
                              numSegmentsPerCurve, csBoundaryConditions, weightingMethod);
                    }

                    double pathWeightLog10
                          = twisty::PathWeighting::WeightCurveViaCurvatureLog10_CudaSafe(
                                &(pPerGlobalThreadScratchSpaceCurvatures
                                            [CurrentThreadCurvatureStartIdx]),
                                (numSegmentsPerCurve - 1), pWeightLookupTable,
                                weightLookupTableSize, ds, minCurvature, maxCurvature,
                                curvatureStepSize);
                    pathWeightLog10 += pathNormalizerLog10;

                    if (pathCount < numPathsToSkipPerThread) {
                        // Skip
                    } else {
                        // Else, contribute to the paths
                        CombinedWeightValues_C_AddValue(
                              perThreadWeightValues[threadIdx.x], pathWeightLog10);
                    }
                }
            }
        }

        // First thread in warp responsible for combining all the weights into one
        __syncthreads();

        if (threadIdx.x == 0) {
            for (uint32_t warpThreadIdx = 1; warpThreadIdx < blockDim.x; ++warpThreadIdx) {
                perThreadWeightValues[0] = CombinedWeightValues_C_CombineValues(
                      perThreadWeightValues[0], perThreadWeightValues[warpThreadIdx]);
            }

            // Finally, we write to the combined final values
            pFinalCombinedValues[blockIdx.x * numCombinedWeightValuesPerWarp
                  + combinedWeightValuesWarpIdx]
                  = perThreadWeightValues[0];
        }
    }
}

}