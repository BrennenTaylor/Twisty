#include "hip/hip_runtime.h"
#include "GpuFullExperimentRunnerGeneral2.h"

#include "CurvePerturbUtils.h"

#include "CurveUtils.h"
#include "MathConsts.h"

#include "Twisty_Cuda_Helpers.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cuda_occupancy.h>
#include <>
#include <hip/hip_math_constants.h>
#include <hip/hip_runtime.h>

#include <omp.h>

#include <assert.h>
#include <ctime>
#include <fstream>
#include <limits>

#include <chrono>
#include <thread>

const uint32_t PerturbBlockSize = 32;

struct GpuVec3
{
    float x;
    float y;
    float z;
};

//#define DetailedPurturb
//#define SingleThreadDebugMode

/*

    In this, we allocate as follows:

    1. Each thread gets its own copy of the initial seeed curve. This is the same for all threads.
    2. Each thread then purturbs into a different curve each time. These all diverge, but run the same algorithm, so performance should be ok.
    3. After each perturb, we weight and generate a single double value for each path. These are then returned to the CPU.

*/

// Cuda Functions
namespace twisty
{
    // Dispatch kernel for initializing the hiprand states
    // This really should be generalized or better thought out for experimentation sake
    __global__ void GpuFullExperimentRunnerGeneral2_InitializeCurandState(uint32_t seed, hiprandState_t *pStates, uint32_t maxNumStates)
    {
        // Unique index
        const uint32_t globalThreadIdx = threadIdx.x + blockIdx.x * blockDim.x;
        if (globalThreadIdx < maxNumStates)
        {
            hiprand_init(seed + globalThreadIdx, 0, 0, &pStates[globalThreadIdx]);
        }
    }


    __global__ void GpuFullExperimentRunnerGeneral2_PerturbControl(
        int32_t numExperimentPaths,
        int32_t numPathsPerBlock,
        int32_t numPathsToSkipPerThread,
        uint32_t numSegmentsPerCurve,
        hiprandState_t* pRandStates,
        float* pPerThreadPositions,
        float* pPerThreadTangents,
        float* pPerThreadCurvatures,

        double* pCachedSegmentWeights,

        double* pPerBlockFinalWeights,
        double* pPerBlockFinalDifferences,

        float segmentLength,
        float scattering,
        float absorbtion,
        double* pLookupTable,
        float minCurvature,
        float maxCurvature,
        float curvatureStepSize
    )
    {
        __shared__ double FinalThreadWeights[PerturbBlockSize];
        __shared__ double FinalThreadDifferences[PerturbBlockSize];
        __shared__ double FinalThreadMaxes[PerturbBlockSize];

        GpuFullExperimentRunnerGeneral2_GeneratePathBatchPerturbations(
            FinalThreadWeights,
            FinalThreadDifferences,
            FinalThreadMaxes,
            numExperimentPaths,
            numPathsPerBlock,
            numPathsToSkipPerThread,
            numSegmentsPerCurve,
            pRandStates,
            pPerThreadPositions,
            pPerThreadTangents,
            pPerThreadCurvatures,

            pCachedSegmentWeights,

            pPerBlockFinalWeights,
            pPerBlockFinalDifferences,

            segmentLength,
            scattering,
            absorbtion,
            pLookupTable,
            minCurvature,
            maxCurvature,
            curvatureStepSize
        );

        __syncthreads();
    }


    // Dispatch which atually runs the purtibation algorithm on the GPU
    __device__ void GpuFullExperimentRunnerGeneral2_GeneratePathBatchPerturbations(
        double* pFinalThreadWeightsShared,
        double* pFinalThreadWeightsDifferences,
        double* pFinalThreadMaxes,
        int32_t numExperimentPaths,
        int32_t numPathsPerBlock,
        int32_t numPathsToSkipPerThread,
        uint32_t numSegmentsPerCurve,
        hiprandState_t* pRandStates, 
        float* pPerThreadPositions,
        float* pPerThreadTangents,
        float* pPerThreadCurvatures,

        double* pCachedSegmentWeights,

        double* pPerBlockFinalWeights,
        double* pPerBlockFinalDifferences,

        float segmentLength,
        float scattering,
        float absorbtion,
        double* pLookupTable,
        float minCurvature,
        float maxCurvature,
        float curvatureStepSize
    )
    {
        int32_t globalThreadIdx = threadIdx.x + blockDim.x * blockIdx.x;
        int32_t numPathsPerThread = (numPathsPerBlock + blockDim.x - 1) / blockDim.x;

        // Check if we are past the experiment
        {
            int32_t globalPathIdx = numPathsPerThread * globalThreadIdx;
            if (globalPathIdx >= numExperimentPaths)
            {
                //printf("Not executing thread idx: %d\n", globalThreadIdx);

                // We dont want to continue if we have already generated the correct number of paths.
                return;
            }
        }

        // The current thread is stored at the beginning
        // We want to index into shared memory via threadIdx.x as this is assigned per block
        const uint32_t NumPosPerCurve = (numSegmentsPerCurve + 1);
        const uint32_t NumTanPerCurve = (numSegmentsPerCurve + 1);
        const uint32_t NumCurvaturesPerCurve = numSegmentsPerCurve;

        // 3 floats per pos and tan
        const uint32_t CurrentThreadPosStartIdx = 3 *  NumPosPerCurve * globalThreadIdx;
        const uint32_t CurrentThreadTanStartIdx = 3 * NumTanPerCurve * globalThreadIdx;
        const uint32_t CurrentThreadCurvatureStartIdx = NumCurvaturesPerCurve * globalThreadIdx;

        float c = scattering + absorbtion;
        float absorbtionConst = std::exp(-c * segmentLength) / (2.0 * HIP_PI_F * HIP_PI_F);
        float absorbtionConstLog10 = std::log10(absorbtionConst);

        const double MaxDoubleLog10 = 300;

        double currentMaxWeightLog10 = 0.0;
        double currentMaxPossibleFinalWeightLog10 = 0.0;
        double currentDifference = 0.0;
        double runningTotalPathBatchLog10 = 0.0;


        double runningSinglePathWeightLog10 = 0.0;
        // Lets precache all the segment weights
        {
            for (uint32_t segIdx = 0; segIdx < numSegmentsPerCurve; ++segIdx)
            {
                float curvature = pPerThreadCurvatures[CurrentThreadCurvatureStartIdx + segIdx];

                float distance = curvature - minCurvature;
                float realIdx = distance / curvatureStepSize;
                uint32_t leftIdx = floor(realIdx);
                uint32_t rightIdx = leftIdx + 1;

                float leftLookup = pLookupTable[leftIdx];
                float rightLookup = pLookupTable[rightIdx];
                float leftDist = distance - (leftIdx * curvatureStepSize);
                double interpolatedResult = leftLookup * (1.0f - leftDist) + (rightLookup * leftDist);
                double interpolatedResultLog10 = std::log(interpolatedResult);
                double segmentWeightLog10 = interpolatedResultLog10;

                segmentWeightLog10 += absorbtionConstLog10;

                // Update the running path weight. We also want to cache the segment weights
                runningSinglePathWeightLog10 += segmentWeightLog10;
                pCachedSegmentWeights[segIdx + (numSegmentsPerCurve * globalThreadIdx)] = segmentWeightLog10;
            }

#if defined(SingleThreadDebugMode)
            {
                printf("Cached Weights:\n");

                for (uint32_t segIdx = 0; segIdx < numSegmentsPerCurve; segIdx++)
                {
                    printf("\tCached Weight: <%0.6f>\n", pCachedSegmentWeights[segIdx + numSegmentsPerCurve * globalThreadIdx]);
                }
            }
#endif

            //if (globalThreadIdx == 1)
            //{
            //    printf("Cached Weights:\n");

            //    for (uint32_t segIdx = 0; segIdx < numSegmentsPerCurve; segIdx++)
            //    {
            //        printf("\tCached Weight: <%0.6f>\n", pCachedSegmentWeights[segIdx + numSegmentsPerCurve * globalThreadIdx]);
            //    }
            //}
        }

        //if (globalThreadIdx == 1)
        //{
        //    printf("Running path weight after cache: %0.6f\n", runningPathWeight);
        //}

        // Now, we can begin the actual algorithm
        {
            // This is the perturbation piece.
            // Can we do this in place

            uint32_t numCurvesInBatch = 0;
            uint32_t outputIdx = 0;

            int32_t cacheStartPathIdx = numPathsPerThread * globalThreadIdx;

            for (int32_t pathCount = 0; pathCount < (numPathsToSkipPerThread + numPathsPerThread); ++pathCount)
            {
                //if (globalThreadIdx == 0)
                //{
                //    printf("Path count: %d\n", pathCount);
                //}

                // Start at the thread's first path idx
                int32_t currentPathIdx = numPathsPerThread * globalThreadIdx + pathCount - numPathsToSkipPerThread;
                //if (globalThreadIdx == 0)
                //{
                //    printf("Current path idx: %d\n", currentPathIdx);
                //}

                if (currentPathIdx >= numExperimentPaths)
                {
#ifdef BlockingOutputThread
                    {
                        std::scoped_lock<std::mutex> lock(outputThreadMutex);
                        std::cout << "Returning, all paths complete" << std::endl;
                    }
#endif

#if defined(ExportPathBatches)
                    if (numCurvesInBatch > 0)
                    {
                        ExportPathBatchesMutex.lock();

                        if (threadIdx == 11)
                        {
                            std::cout << "Should be exporting thread 12" << std::endl;
                        }


                        curvesMetadataFile << threadIdx << " ";
                        curvesMetadataFile << outputIdx << " ";
                        curvesMetadataFile << numCurvesInBatch << std::endl;

                        curvesBinaryFile.write((char*)pathBatchCache.data(), sizeof(Farlor::Vector3) * NumPosPerCurve * numCurvesInBatch);
                        numCurvesInBatch = 0;
                        outputIdx++;

                        ExportPathBatchesMutex.unlock();
                    }
#endif


                    // We dont want to continue if we have already generated the correct number of paths.
                    //if (globalThreadIdx == 0)
                    //{
                    //    printf("Exiting, we generated the correct number of paths\n");
                    //}
                    break;
                }

                //if (globalThreadIdx == 0)
                //{
                //    printf("Beginning perturb of path %d\n", pathCount);
                //}

                // Do the perturb now
                {
                    // We bound on left by one as we dont want to rotate the first segment at all
                    // Left bound by m-2 as we at least want there to be one point between the left and right points selected so an actual perturbation occurs
                    float leftPtRand = hiprand_uniform(&pRandStates[globalThreadIdx]);
                    float rightPtRand = hiprand_uniform(&pRandStates[globalThreadIdx]);

                    //int32_t leftPointIndex = 2;
                    //int32_t rightPointIndex = 4;

                    unsigned int leftPointIndex = floorf(leftPtRand * ((numSegmentsPerCurve - 3) - 1) + 1);
                    unsigned int rightPointIndex = floorf(leftPtRand * ((numSegmentsPerCurve - 1) - (leftPointIndex + 2)) + (leftPointIndex + 2));

#if defined(SingleThreadDebugMode)
                    {
                        printf("Left point idx: %d\n", leftPointIndex);
                        printf("Right point idx: %d\n", rightPointIndex);
                    }
#endif

                    assert(leftPointIndex < rightPointIndex);
                    assert((rightPointIndex - leftPointIndex) >= 2);

                    // We need two frames for each segment to get the new curvature and torsion.
                    // we need the frame left of the segment, as well as the frame right of the segment.

                    // The left point also will act as the origin for rotating the points between leftPoint and rightPoint
                    float* pLeftPoint = pPerThreadPositions + CurrentThreadPosStartIdx  + PositionFloatCount * leftPointIndex;
                    float* pRightPoint = pPerThreadPositions + CurrentThreadPosStartIdx + PositionFloatCount * rightPointIndex;
                    
#if defined(SingleThreadDebugMode)
                    {
                        printf("Left Point: (%.6f, %.6f, %.6f)\n", pLeftPoint[0], pLeftPoint[1], pLeftPoint[2]);
                        printf("Right Point: (%.6f, %.6f, %.6f)\n", pRightPoint[0], pRightPoint[1], pRightPoint[2]);
                    }
#endif

                    float axisOfRotation[3];
                    axisOfRotation[0] = pRightPoint[0] - pLeftPoint[0];
                    axisOfRotation[1] = pRightPoint[1] - pLeftPoint[1];
                    axisOfRotation[2] = pRightPoint[2] - pLeftPoint[2];

                    NormalizeVector3f(axisOfRotation);


#if defined(SingleThreadDebugMode)
                    {
                        printf("Axis of rotation: (%.6f, %.6f, %.6f)\n", axisOfRotation[0], axisOfRotation[1], axisOfRotation[2]);
                    }
#endif

                    //if (globalThreadIdx == 1)
                    //{
                    //    printf("Axis before (%.6f, %.6f, %.6f)\n",
                    //        axisOfRotation[0], axisOfRotation[1], axisOfRotation[2]
                    //    );
                    //}

                    //float randomAngle = 1.38f;
                    float randomAngle = (hiprand_uniform(&pRandStates[globalThreadIdx]) * 360.0f) - 180.0f;

#if defined(SingleThreadDebugMode)
                    {
                        printf("randomAngle: %.6f\n", randomAngle);
                    }
#endif


                    float rotationMatrix[9];
                    RotationMatrixAroundAxis(randomAngle, axisOfRotation, rotationMatrix);
#if defined(SingleThreadDebugMode)
                    {
                        printf("Rotation Matrix\n\t(%.6f, %.6f, %.6f)\n\t(%.6f, %.6f, %.6f)\n\t(%.6f, %.6f, %.6f)\n",
                            rotationMatrix[0], rotationMatrix[1], rotationMatrix[2],
                            rotationMatrix[3], rotationMatrix[4], rotationMatrix[5],
                            rotationMatrix[6], rotationMatrix[7], rotationMatrix[8]
                        );
                    }
#endif

                    //if (globalThreadIdx == 1)
                    //{
                    //    printf("Normalized axis(%.6f, %.6f, %.6f)\n",
                    //        axisOfRotation[0], axisOfRotation[1], axisOfRotation[2]
                    //    );

                    //    printf("Rotation Matrix\n\t(%.6f, %.6f, %.6f)\n\t(%.6f, %.6f, %.6f)\n\t(%.6f, %.6f, %.6f)\n",
                    //        rotationMatrix[0], rotationMatrix[1], rotationMatrix[2],
                    //        rotationMatrix[3], rotationMatrix[4], rotationMatrix[5],
                    //        rotationMatrix[6], rotationMatrix[7], rotationMatrix[8]
                    //    );
                    //}

                    uint32_t numChanged = 0;
                    for (uint32_t pointIdx = (leftPointIndex + 1); pointIdx < rightPointIndex; ++pointIdx)
                    {
                        numChanged++;
                        float* pCurrentPoint = pPerThreadPositions + CurrentThreadPosStartIdx + PositionFloatCount * pointIdx;

                        float shiftedPoint[3];
                        shiftedPoint[0] = pCurrentPoint[0] - pLeftPoint[0];
                        shiftedPoint[1] = pCurrentPoint[1] - pLeftPoint[1];
                        shiftedPoint[2] = pCurrentPoint[2] - pLeftPoint[2];
                        // Rotate and stuff back in shifted point
                        RotateVectorByMatrix(rotationMatrix, shiftedPoint);

                        // Update the point with the rotated version
                        pCurrentPoint[0] = shiftedPoint[0] + pLeftPoint[0];
                        pCurrentPoint[1] = shiftedPoint[1] + pLeftPoint[1];
                        pCurrentPoint[2] = shiftedPoint[2] + pLeftPoint[2];
                    }

                    //Now, simply compute the difference in positions at the two edges of the rotated rigidbody.
                    //We can do a different approach later.

                    // Left side
                    {
                        float* pLeftPointTanCalc = pPerThreadPositions + CurrentThreadPosStartIdx + PositionFloatCount * leftPointIndex;
                        float* pRightPointTanCalc = pPerThreadPositions + CurrentThreadPosStartIdx + PositionFloatCount * (leftPointIndex + 1);

                        //if (globalThreadIdx == 1)
                        //{
                        //    printf("Left Tangent Left Point: (%.6f, %.6f, %.6f)\n", pLeftPointTanCalc[0], pLeftPointTanCalc[1], pLeftPointTanCalc[2]);
                        //    printf("Left Tangent Right Point: (%.6f, %.6f, %.6f)\n", pRightPointTanCalc[0], pRightPointTanCalc[1], pRightPointTanCalc[2]);
                        //}

                        float* pCurrentTan = pPerThreadTangents + CurrentThreadTanStartIdx + TangentFloatCount * leftPointIndex;

                        pCurrentTan[0] = pRightPointTanCalc[0] - pLeftPointTanCalc[0];
                        pCurrentTan[1] = pRightPointTanCalc[1] - pLeftPointTanCalc[1];
                        pCurrentTan[2] = pRightPointTanCalc[2] - pLeftPointTanCalc[2];

                        NormalizeVector3f(pCurrentTan);

                        //if (globalThreadIdx == 1)
                        //{
                        //    printf("New Left Tangent: (%.6f, %.6f, %.6f)\n", pCurrentTan[0], pCurrentTan[1], pCurrentTan[2]);
                        //}
                    }

                    // Right side
                    {
                        float* pLeftPointTanCalc = pPerThreadPositions + CurrentThreadPosStartIdx + PositionFloatCount * (rightPointIndex - 1);
                        float* pRightPointTanCalc = pPerThreadPositions + CurrentThreadPosStartIdx + PositionFloatCount * (rightPointIndex);

                        float* pCurrentTan = pPerThreadTangents + CurrentThreadTanStartIdx + TangentFloatCount * (rightPointIndex - 1);

                        pCurrentTan[0] = pRightPointTanCalc[0] - pLeftPointTanCalc[0];
                        pCurrentTan[1] = pRightPointTanCalc[1] - pLeftPointTanCalc[1];
                        pCurrentTan[2] = pRightPointTanCalc[2] - pLeftPointTanCalc[2];

                        NormalizeVector3f(pCurrentTan);

                        //if (globalThreadIdx == 1)
                        //{
                        //    printf("New Right Tangent: (%.6f, %.6f, %.6f)\n", pCurrentTan[0], pCurrentTan[1], pCurrentTan[2]);
                        //}
                    }

                    // Update left curvature
                    {
                        float* pLeftTanCurvatureCalc = pPerThreadTangents + CurrentThreadTanStartIdx + TangentFloatCount * (leftPointIndex - 1);
                        float* pRightTanCurvatureCalc = pPerThreadTangents + CurrentThreadTanStartIdx + TangentFloatCount * leftPointIndex;
                        float* pCurvature = pPerThreadCurvatures + CurrentThreadCurvatureStartIdx + (leftPointIndex - 1);


                        float temp[3];
                        temp[0] = (pRightTanCurvatureCalc[0] - pLeftTanCurvatureCalc[0]) * (1.0f / segmentLength);
                        temp[1] = (pRightTanCurvatureCalc[1] - pLeftTanCurvatureCalc[1]) * (1.0f / segmentLength);
                        temp[2] = (pRightTanCurvatureCalc[2] - pLeftTanCurvatureCalc[2]) * (1.0f / segmentLength);

                        float curvature = MagVector3f(temp);
                        pCurvature[0] = curvature;

                        //if (globalThreadIdx == 1)
                        //{
                        //    printf("Left Segment Curvature: %.6f\n", curvature);
                        //}

                        // Also, cache the weight of that changed segment
                        float distance = curvature - minCurvature;
                        float realIdx = distance / curvatureStepSize;
                        uint32_t leftIdx = floor(realIdx);
                        uint32_t rightIdx = leftIdx + 1;

                        double leftLookup = pLookupTable[leftIdx];
                        double rightLookup = pLookupTable[rightIdx];

                        float leftDist = distance - (leftIdx * curvatureStepSize);

                        double interpolatedResult = leftLookup * (1.0f - leftDist) + (rightLookup * leftDist);
                        double interpolatedResultLog10 = std::log10(interpolatedResult);
                        double segmentWeightLog10 = interpolatedResultLog10;
                        segmentWeightLog10 += absorbtionConstLog10;

                        //if (globalThreadIdx == 1)
                        //{
                        //    printf("Left Segment Weight: %.6f\n", segmentWeight);
                        //}

                        // Remove old segmentWeight
                        runningSinglePathWeightLog10 -= pCachedSegmentWeights[(leftPointIndex - 1) + (numSegmentsPerCurve * globalThreadIdx)];
                        pCachedSegmentWeights[(leftPointIndex - 1) + (numSegmentsPerCurve * globalThreadIdx)] = segmentWeightLog10;
                        runningSinglePathWeightLog10 += segmentWeightLog10;

                        //if (globalThreadIdx == 1)
                        //{
                        //    printf("Running path weight after left: %.6f\n", runningPathWeight);
                        //}
                    }

                    // Update right curvature
                    {
                        float* pLeftTanCurvatureCalc = pPerThreadTangents + CurrentThreadTanStartIdx + TangentFloatCount * (rightPointIndex - 1);
                        float* pRightTanCurvatureCalc = pPerThreadTangents + CurrentThreadTanStartIdx + TangentFloatCount * rightPointIndex;
                        float* pCurvature = pPerThreadCurvatures + CurrentThreadCurvatureStartIdx + (rightPointIndex - 1);

                        //if (globalThreadIdx == 1)
                        //{
                        //    printf("Rights Calc Left Tangent: (%.6f, %.6f, %.6f)\n", pLeftTanCurvatureCalc[0], pLeftTanCurvatureCalc[1], pLeftTanCurvatureCalc[2]);
                        //    printf("Rights Calc Right Tangent: (%.6f, %.6f, %.6f)\n", pRightTanCurvatureCalc[0], pRightTanCurvatureCalc[1], pRightTanCurvatureCalc[2]);
                        //}

                        float temp[3];
                        temp[0] = (pRightTanCurvatureCalc[0] - pLeftTanCurvatureCalc[0]) * (1.0f / segmentLength);
                        temp[1] = (pRightTanCurvatureCalc[1] - pLeftTanCurvatureCalc[1]) * (1.0f / segmentLength);
                        temp[2] = (pRightTanCurvatureCalc[2] - pLeftTanCurvatureCalc[2]) * (1.0f / segmentLength);

                        float curvature = MagVector3f(temp);
                        pCurvature[0] = curvature;

                        //if (globalThreadIdx == 1)
                        //{
                        //    printf("Right Segment Curvature: %.6f\n", curvature);
                        //}

                        // Also, cache the weight of that changed segment
                        float distance = curvature - minCurvature;
                        float realIdx = distance / curvatureStepSize;
                        uint32_t leftIdx = floor(realIdx);
                        uint32_t rightIdx = leftIdx + 1;

                        double leftLookup = pLookupTable[leftIdx];
                        double rightLookup = pLookupTable[rightIdx];

                        float leftDist = distance - (leftIdx * curvatureStepSize);

                        double interpolatedResult = leftLookup * (1.0f - leftDist) + (rightLookup * leftDist);
                        double interpolatedResultLog10 = std::log10(interpolatedResult);
                        double segmentWeightLog10 = interpolatedResultLog10;
                        segmentWeightLog10 += absorbtionConstLog10;

                        //if (globalThreadIdx == 1)
                        //{
                        //    printf("Right Segment Weight: %.6f\n", segmentWeight);
                        //}

                        // Remove old segmentWeight
                        runningSinglePathWeightLog10 -= pCachedSegmentWeights[(rightPointIndex - 1) + (numSegmentsPerCurve * globalThreadIdx)];
                        pCachedSegmentWeights[(rightPointIndex - 1) + (numSegmentsPerCurve * globalThreadIdx)] = segmentWeightLog10;
                        runningSinglePathWeightLog10 += segmentWeightLog10;

                        //if (globalThreadIdx == 1)
                        //{
                        //    printf("Running path weight after right: %.6f\n", runningPathWeight);
                        //}
                    }

                    /*if (globalThreadIdx == 0)
                    {
                        printf("Made it to the end of path %d\n", pathCount);
                    }*/

                    if (pathCount < numPathsToSkipPerThread)
                    {
                        // Skip
                    }
                    //else
                    //{
                    //    // Else, contribute to the paths
                    //    //int32_t currentPathIdx = numPathsPerThread * globalThreadIdx + pathCount - numPathsToSkipPerThread;
                    //    assert(currentPathIdx >= numPathsPerThread * globalThreadIdx);
                    //    pCompressedPathWeights[currentPathIdx] = runningPathWeight;
                    //    //if (globalThreadIdx == 0)
                    //    //{
                    //    //    printf("Updating path weight at %d to %0.6f\n", currentPathIdx, runningPathWeight);
                    //    //}
                    //}
                    else
                    {
                        // Else, contribute to the paths
                        int32_t currentPathIdx = numPathsPerThread * globalThreadIdx + pathCount - numPathsToSkipPerThread;
                        assert(currentPathIdx >= numPathsPerThread * globalThreadIdx);

                        int32_t pathNumber = pathCount - numPathsToSkipPerThread;

                        if (pathNumber == 0)
                        {
                            currentMaxWeightLog10 = runningSinglePathWeightLog10;
                            currentMaxPossibleFinalWeightLog10 = runningSinglePathWeightLog10 + 6.0;
                            currentDifference = MaxDoubleLog10 - currentMaxPossibleFinalWeightLog10;
                            runningTotalPathBatchLog10 += pow(10.0, runningSinglePathWeightLog10 + currentDifference);
                        }
                        else
                        {
                            double newMaxWeightLog10 = runningSinglePathWeightLog10;
                            // If this checks out, we have the same maximum and thus can just adjust things up
                            if (currentMaxWeightLog10 > newMaxWeightLog10)
                            {
                                runningTotalPathBatchLog10 += pow(10.0, runningSinglePathWeightLog10 + currentDifference);
                                continue;
                            }

                            // If we are past, then we have a new maximum and need to adjust
                            // New difference
                            double newMaxPossibleFinalWeightLog10 = newMaxWeightLog10 + 6.0;
                            double newDifference = MaxDoubleLog10 - newMaxPossibleFinalWeightLog10;

                            double differenceDelta = newDifference - currentDifference;
                            double log10RunningTotal = std::log10(runningTotalPathBatchLog10);
                            double adjustedLog10RunningTotal = log10RunningTotal + differenceDelta;
                            runningTotalPathBatchLog10 = std::pow(10.0, adjustedLog10RunningTotal);

                            // Update
                            currentMaxWeightLog10 = newMaxWeightLog10;
                            currentMaxPossibleFinalWeightLog10 = newMaxPossibleFinalWeightLog10;
                            currentDifference = newDifference;

                            runningTotalPathBatchLog10 += pow(10.0, runningSinglePathWeightLog10 + currentDifference);
                        }

                        if (threadIdx.x == 0)
                        {
                            pPerBlockFinalWeights[blockIdx.x] = runningTotalPathBatchLog10;
                            pPerBlockFinalDifferences[blockIdx.x] = currentDifference;
                        }
                    }
                }
            }
        }
    }
}

namespace twisty
{

    GpuFullExperimentRunnerGeneral2::GpuFullExperimentRunnerGeneral2(ExperimentRunner::ExperimentParameters& experimentParams,
        Bootstrapper& bootstrapper)
        : ExperimentRunner(experimentParams, bootstrapper)
        , m_rng()
        , m_numSMs(0)
        , m_warpSize(0)
        , m_maxThreadsPerMultiprocessor(0)
        , m_pPerGlobalThreadRandStates(nullptr)
        , m_pPerBlockFinalWeights(nullptr)
        , m_pPerBlockFinalDifferences(nullptr)
    {
        uint32_t seed = m_experimentParams.curvePurturbSeed;
        if (seed == 0)
        {
            seed = time(0);
        }
        std::cout << "\nPurturb seed used: " << seed << std::endl;
        m_rng = std::mt19937(seed);
    }


    GpuFullExperimentRunnerGeneral2::~GpuFullExperimentRunnerGeneral2()
    {
    }

    bool GpuFullExperimentRunnerGeneral2::Setup()
    {
        bool result = SetupCudaDevice();
        if (!result)
        {
            printf("Failed to setup cuda device\n");
            return false;
        }

        // Ask the bootstrapper to generate a discrete curve.
        // If we fail, we want to exit the experiment.

        bool successfulGen = false;
        while (!successfulGen)
        {
            m_upInitialCurve = m_bootstrapper.CreateCurve(m_experimentParams.numSegmentsPerCurve);
            if (!m_upInitialCurve)
            {
                printf("Failed to create bootstrap curve.\n");
                return false;
            }

            // Lets also get the error of the initial curve, just to know
            float curveError = CurveUtils::CalculateCurveError(*m_upInitialCurve);
            std::cout << "\tSeed curve error: " << curveError << std::endl;

            if (curveError < m_experimentParams.maximumBootstrapCurveError)
            {
                successfulGen = true;
            }
        }

        return true;
    }



    // This sets up the cuda device for use. This could be pulled out into a more general class.
    bool GpuFullExperimentRunnerGeneral2::SetupCudaDevice()
    {
        // Query for the number of devices avalible
        int32_t numDevices = 0;
        CudaSafeErrorCheck(hipGetDeviceCount(&numDevices), "Get Devices");

        // We need at least one cuda device
        if (numDevices <= 0)
        {
            printf("No CUDA device avalible. Cannot execute experiment.\n");
            return false;
        }

        // Iterate over all devices and report the device stats.
        for (int32_t i = 0; i < numDevices; ++i)
        {
            hipDeviceProp_t prop;
            CudaSafeErrorCheck(hipGetDeviceProperties(&prop, i), "Get Device Prop");
            printf("\nDevice Number: %d\n", i);
            printf("\tDevice name: %s\n", prop.name);
            printf("\tSM Count: %d\n", prop.multiProcessorCount);
            printf("\tSM Shared Memory: %d\n", prop.sharedMemPerBlock);
            printf("\tWarp Size: %d\n", prop.warpSize);
            printf("\tThreads per SM: %d\n", prop.maxThreadsPerMultiProcessor);
            printf("\tPeak Memory Bandwidth (GB/s): %f\n", 2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
        }

        // We select the first device only
        const uint32_t selectedDeviceIdx = 0;
        std::cout << "\nSelected device: " << selectedDeviceIdx << std::endl;
        hipDeviceProp_t deviceProp;
        CudaSafeErrorCheck(hipGetDeviceProperties(&deviceProp, 0), "Get first device prop");

        m_numSMs = deviceProp.multiProcessorCount;
        m_warpSize = deviceProp.warpSize;
        m_maxThreadsPerMultiprocessor = deviceProp.maxThreadsPerMultiProcessor;

        return true;
    }

    // We calculate the dispatch parameters based off kernel/problem complexity
    //bool GpuFullExperimentRunnerGeneral2::SetupKernelDispatchParameters(uint32_t numPathWeightsInShared, uint32_t& numGlobalPerturbThreads, uint32_t& perturbBlockSize, uint32_t& perturbGridSize)
    //{
    //    //// Calculate minimum grid size and block size required to achieves maximum potential occupancy for GeneratePathBatchPutrubations
    //    //{
    //    //    int blockSizePurturbKernel = 0;
    //    //    int minGridSizePurturbKernel = 0;
    //    //    size_t maxBlockSize = 0;

    //    //    size_t segmentsPerCurve = m_experimentParams.numSegmentsPerCurve;

    //    //    size_t sharedMemoryUse = 0;
    //    //    
    //    //    //// https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__HIGHLEVEL.html#group__CUDART__HIGHLEVEL_1g77b3bfb154b86e215a5bc01509ce8ea6
    //    //    CudaSafeErrorCheck(hipOccupancyMaxPotentialBlockSize(&minGridSizePurturbKernel, &blockSizePurturbKernel,
    //    //        GpuFullExperimentRunnerGeneral2_GeneratePathBatchPutrubations, sharedMemoryUse, maxBlockSize),
    //    //        "hipOccupancyMaxPotentialBlockSize for perturb kernel");

    //    //    std::cout << "\nGeneratePutrubations: " << std::endl;
    //    //    std::cout << "\tRecommended Block Size: " << blockSizePurturbKernel << std::endl;
    //    //    std::cout << "\tMin Grid Size For Max Occupency: " << minGridSizePurturbKernel << std::endl;

    //    //    // Assume we generate one path per thread
    //    //    perturbGridSize = minGridSizePurturbKernel;
    //    //    perturbBlockSize = blockSizePurturbKernel;

    //    //    // We hardcode this for some reason...?
    //    //    perturbGridSize = 1024 * 2 * 2;
    //    //    perturbBlockSize = Pert;

    //    //    numGlobalPerturbThreads = perturbGridSize * perturbBlockSize;

    //    //    printf("\tGeneratePathBatchPutrubations Grid Size: %d\n", perturbGridSize);
    //    //    printf("\tGeneratePutrubations Block Size: %d\n", perturbBlockSize);
    //    //    printf("\tGeneratePutrubations Num Block Threads: %d\n", numGlobalPerturbThreads);
    //    //}

    //    return true;
    //}

    bool GpuFullExperimentRunnerGeneral2::SetupCuRandStates(uint32_t numGlobalPerturbThreads)
    {
        // Random Seed Kernel
        // Every block thread needs its own hiprand state
        CudaSafeErrorCheck(hipMalloc((void**)&m_pPerGlobalThreadRandStates, sizeof(hiprandState_t) * numGlobalPerturbThreads),
            "RandState malloc");
        
        int blockSizeRandKernel = 0;
        int minGridSizeRandKernel = 0;
        size_t sharedMemoryUse = 0;
        size_t maxBlockSize = 0;
        hipOccupancyMaxPotentialBlockSize(&minGridSizeRandKernel, &blockSizeRandKernel, GpuFullExperimentRunnerGeneral2_InitializeCurandState, sharedMemoryUse, maxBlockSize);
        std::cout << "\nInitializeCurandState: " << std::endl;
        std::cout << "\tBlock Size: " << blockSizeRandKernel << std::endl;
        std::cout << "\tMin Grid Size: " << minGridSizeRandKernel << std::endl;
        
        size_t gridSizeRandKernel = ((numGlobalPerturbThreads) + blockSizeRandKernel - 1) / blockSizeRandKernel;

        printf("\tInitializeCurandState Grid Size: %d\n", gridSizeRandKernel);
        printf("\tInitializeCurandState Block Size: %d\n", blockSizeRandKernel);
        
        // Dispatch CurandState
        // We need a dispatch that initializes hiprand per thread
        {
            dim3 gridSize(gridSizeRandKernel, 1, 1);
            dim3 blockSize(blockSizeRandKernel, 1, 1);
            size_t sharedMemorySizeBytes = 0;
            hipStream_t stream = 0;

            GpuFullExperimentRunnerGeneral2_InitializeCurandState <<<gridSize, blockSize, sharedMemorySizeBytes, stream >>>(
                static_cast<uint32_t>(m_experimentParams.curvePurturbSeed),
                m_pPerGlobalThreadRandStates,
                numGlobalPerturbThreads
            );

            CudaSafeErrorCheck(hipGetLastError(), "Rand state init kernal launch");
            CudaSafeErrorCheck(hipDeviceSynchronize(), "Rand state kernel sync");
        }

        return true;
    }


    bool GpuFullExperimentRunnerGeneral2::SetupCrossDispatchCurveData(uint32_t perturbGridSize, uint32_t perturbBlockSize)
    {
        const uint32_t numGlobalPerturbThreads = perturbGridSize * perturbBlockSize;

        std::vector<Farlor::Vector3> initialCurvePositions(m_experimentParams.numSegmentsPerCurve + 1);
        std::vector<Farlor::Vector3> initialCurveTangents(m_experimentParams.numSegmentsPerCurve + 1);
        std::vector<float> initialCurveCurvatures(m_experimentParams.numSegmentsPerCurve);

        // Positions
        // Hard code the first two positions
        initialCurvePositions[0] = m_upInitialCurve->m_basePos;
        initialCurvePositions[1] = m_upInitialCurve->m_basePos + m_upInitialCurve->m_baseTangent.Normalized() * m_upInitialCurve->m_segmentLength;
        for (uint32_t segmentIdx = 2; segmentIdx < m_experimentParams.numSegmentsPerCurve; ++segmentIdx)
        {
            initialCurvePositions[segmentIdx] = m_upInitialCurve->m_positions[segmentIdx];
        }
        // Hard code the final position
        initialCurvePositions[m_experimentParams.numSegmentsPerCurve] = m_upInitialCurve->m_targetPos;

#if defined(DetailedPurturb)
        {
            std::cout << "Positions" << std::endl;
            for (uint32_t segmentIdx = 0; segmentIdx < m_experimentParams.numSegmentsPerCurve; ++segmentIdx)
            {
                std::cout << "\t" << initialCurvePositions[segmentIdx] << std::endl;
            }
        }
#endif

        // Tangents
        // Hardcode intial tangent
        initialCurveTangents[0] = m_upInitialCurve->m_baseTangent;
        for (uint32_t tanIdx = 1; tanIdx < m_experimentParams.numSegmentsPerCurve; ++tanIdx)
        {
            Farlor::Vector3 leftPos = initialCurvePositions[tanIdx];
            Farlor::Vector3 rightPos = initialCurvePositions[tanIdx + 1];

            initialCurveTangents[tanIdx] = (rightPos - leftPos).Normalized();
        }
        // Final Tangents
        initialCurveTangents[m_experimentParams.numSegmentsPerCurve] = m_upInitialCurve->m_targetTangent;

#if defined(DetailedPurturb)
        {
            std::cout << "Tangents" << std::endl;
            for (uint32_t tanIdx = 0; tanIdx < m_experimentParams.numSegmentsPerCurve; ++tanIdx)
            {
                std::cout << "\t" << initialCurveTangents[tanIdx] << std::endl;
            }
        }
#endif

        // Curvatures
        float segLength = m_upInitialCurve->m_arclength / m_upInitialCurve->m_numSegments;
        for (uint32_t curvatureIdx = 0; curvatureIdx < m_experimentParams.numSegmentsPerCurve; ++curvatureIdx)
        {
            Farlor::Vector3 tanLeft = initialCurveTangents[curvatureIdx];
            Farlor::Vector3 tanRight = initialCurveTangents[curvatureIdx + 1];

            Farlor::Vector3 curvatureVec = (tanRight - tanLeft) * (1.0f / segLength);
            float curvature = curvatureVec.Magnitude();
            initialCurveCurvatures[curvatureIdx] = curvature;
        }

#if defined(DetailedPurturb)
        {
            std::cout << "Curvatures" << std::endl;
            for (uint32_t segmentIdx = 0; segmentIdx < m_experimentParams.numSegmentsPerCurve; ++segmentIdx)
            {
                std::cout << "\t" << initialCurveCurvatures[segmentIdx] << std::endl;
            }
        }
#endif

        const uint32_t NumPosPerCurve = (m_upInitialCurve->m_numSegments + 1);
        const uint32_t NumTanPerCurve = (m_upInitialCurve->m_numSegments + 1);
        const uint32_t NumCurvaturePerCurve = (m_upInitialCurve->m_numSegments);

        CudaSafeErrorCheck(hipMalloc((void**)&m_pPerThreadPositions, sizeof(Farlor::Vector3) * (m_experimentParams.numSegmentsPerCurve + 1) * numGlobalPerturbThreads), "Failed to allocate per thread position memory");
        CudaSafeErrorCheck(hipMalloc((void**)&m_pPerThreadTangents, sizeof(Farlor::Vector3) * (m_experimentParams.numSegmentsPerCurve + 1) * numGlobalPerturbThreads), "Failed to allocate per thread tangent memory");
        CudaSafeErrorCheck(hipMalloc((void**)&m_pPerThreadCurvatures, sizeof(float) * (m_experimentParams.numSegmentsPerCurve) * numGlobalPerturbThreads), "Failed to allocate per thread curvature memory");

        for (uint32_t threadIdx = 0; threadIdx < numGlobalPerturbThreads; ++threadIdx)
        {
            {
                float* pDest = m_pPerThreadPositions + 3 * (m_experimentParams.numSegmentsPerCurve + 1) * threadIdx;
                CudaSafeErrorCheck(hipMemcpy(pDest, initialCurvePositions.data(), sizeof(Farlor::Vector3) * (m_experimentParams.numSegmentsPerCurve + 1), hipMemcpyHostToDevice),
                    "Copy initial curve to device");
            }

            {
                float* pDest = m_pPerThreadTangents + 3 * (m_experimentParams.numSegmentsPerCurve + 1) * threadIdx;
                CudaSafeErrorCheck(hipMemcpy(pDest, initialCurveTangents.data(), sizeof(Farlor::Vector3) * (m_experimentParams.numSegmentsPerCurve + 1), hipMemcpyHostToDevice),
                    "Copy initial curve to device");
            }

            {
                float* pDest = m_pPerThreadCurvatures + 1 * m_experimentParams.numSegmentsPerCurve * threadIdx;
                CudaSafeErrorCheck(hipMemcpy(pDest, initialCurveCurvatures.data(), sizeof(float) * m_experimentParams.numSegmentsPerCurve, hipMemcpyHostToDevice),
                    "Copy initial curve to device");
            }
        }

        // Allocate the segment weight cache
        // TODO: Move this over to shared memory, this should fit
        CudaSafeErrorCheck(hipMalloc((void**)&m_pPerThreadSegmentWeightCache, sizeof(double) * m_experimentParams.numSegmentsPerCurve * numGlobalPerturbThreads), "Failed to allocate cached segment weight buffer");
        
        // Allocate path weighting stuff
        // TODO: Investigate moving this over to shared memory
        CudaSafeErrorCheck(hipMalloc((void**)&m_pPerBlockFinalWeights, sizeof(double) * perturbGridSize), "Failed to allocate block weight buffer");
        CudaSafeErrorCheck(hipMalloc((void**)&m_pPerBlockFinalDifferences, sizeof(double)* perturbGridSize), "Failed to allocate global path weight buffer");

        return true;
    }

    void GpuFullExperimentRunnerGeneral2::CleanupCudaMemory()
    {
        hipFree(m_pPerBlockFinalDifferences);
        hipFree(m_pPerBlockFinalWeights);
        hipFree(m_pPerThreadSegmentWeightCache);
        hipFree(m_pPerThreadCurvatures);
        hipFree(m_pPerThreadTangents);
        hipFree(m_pPerThreadPositions);
        hipFree(m_pWeightLookupTable);
        hipFree(m_pPerGlobalThreadRandStates);
    }

    bool GpuFullExperimentRunnerGeneral2::SetupWeightLookupTexture(const twisty::PathWeighting::WeightLookupTableIntegral& lookupEvaluator)
    {
        auto& weightValues = lookupEvaluator.AccessLookupTable();

        CudaSafeErrorCheck(
            hipMalloc((void**)&m_pWeightLookupTable, sizeof(double) * weightValues.size()),
            "Failed to allocate weight lookup table"
        );

        CudaSafeErrorCheck(
            hipMemcpy(m_pWeightLookupTable, weightValues.data(), sizeof(double) * weightValues.size(), hipMemcpyHostToDevice),
            "Copy weight lookup table to device"
        );

        return true;
    }

    ExperimentRunner::ExperimentResults GpuFullExperimentRunnerGeneral2::RunExperiment()
    {
        const uint32_t numCachedPathWeightsShardMem = 0;

        auto runExperimentTimeStart = std::chrono::high_resolution_clock::now();

        // Calculate grid and block sizes based on the kernels we will call
        // Hardcoded constraint from the double
        const uint32_t blockPathCount = 1000000;
        const uint32_t perturbGridSize = (m_experimentParams.numPathsInExperiment + blockPathCount - 1) / blockPathCount;
        const uint32_t numGlobalPerturbThreads = perturbGridSize * PerturbBlockSize;
        std::cout << "numPathsInExperiment: " << m_experimentParams.numPathsInExperiment << std::endl;
        std::cout << "numPathsPerBatch: " << blockPathCount << std::endl;
        std::cout << "Perturb Block Size required: " << PerturbBlockSize << std::endl;
        std::cout << "Perturb Grid Size required: " << perturbGridSize << std::endl;

        bool result = true;
        auto setupCurandTimeStart = std::chrono::high_resolution_clock::now();
        {
            result = SetupCuRandStates(numGlobalPerturbThreads);
            if (!result)
            {
                printf("Failed to setup hiprand states\n");
                return {};
            }
        }
        auto setupCurandTimeEnd = std::chrono::high_resolution_clock::now();


        /* ---------------------------- */
        auto setupCurveDataStructuresTimeStart = std::chrono::high_resolution_clock::now();
        uint32_t numFailures = 0;
        uint32_t totalFailures = 0;
        uint32_t totalSuccess = 0;
        double ds = m_upInitialCurve->m_arclength / m_experimentParams.numSegmentsPerCurve;
        twisty::PathWeighting::WeightLookupTableIntegral lookupEvaluator(m_experimentParams.weightingParameters, ds);
        

        double minCurvature = 0.0;
        double maxCurvature = 0.0;
        twisty::PathWeighting::CalcMinMaxCurvature(minCurvature, maxCurvature, ds);
        const float curvatureStepSize = (maxCurvature - minCurvature) / m_experimentParams.weightingParameters.numCurvatureSteps;

        // Do non dispatch specific setup
        {
            // Allocate and copy lookup table over to GPU
            result = SetupWeightLookupTexture(lookupEvaluator);
            if (!result)
            {
                printf("Failed to setup weight lookup texture\n");
                return {};
            }

            result = SetupCrossDispatchCurveData(perturbGridSize, PerturbBlockSize);
            if (!result)
            {
                printf("Failed to setup curve device data structures\n");
                return {};
            }
        }
        auto setupCurveDataStructuresTimeEnd = std::chrono::high_resolution_clock::now();
        /* ---------------------------- */

        uint32_t numPathsGenerated = 0;

        // We need to calculate the absorbtion/scattering piece
        boost::multiprecision::cpp_dec_float_100 bigTotalExperimentWeight = 0.0;

        // We need a number of dispatches
        long long perturbTimeCount = 0;
        long long weightCopyTimeCount = 0;
        long long weightCalcTimeCount = 0;

        {
            auto perturbPhaseTimeStart = std::chrono::high_resolution_clock::now();
            {

                // At this point, we know how many curves we'll want to generate. So, we setup our parameters to handle this.
                std::cout << "Experiment Information: " << std::endl;

                //uint32_t numPathsPerThread = (pathsInDispatch + numGlobalPerturbThreads - 1) / numGlobalPerturbThreads;
                //std::cout << "\tNum paths generated per global thread: " << numPathsPerThread << std::endl;


                // Dispatch CurandState
                // We need a dispatch that initializes hiprand per thread
                {
                    // Lets hardcode this for now
                    dim3 dispatchBlockSize(32, 1, 1);
                    dim3 dispatchGridSize(perturbGridSize, 1, 1);
                    // This will change as we start using shared memory
                    size_t sharedMemorySizeBytes = 0;
                    hipStream_t stream = 0;

                    GpuFullExperimentRunnerGeneral2_PerturbControl<<<dispatchGridSize, dispatchBlockSize, 0, stream>>>(
                        m_experimentParams.numPathsInExperiment,
                        blockPathCount,
                        m_experimentParams.numPathsToSkip,
                        m_experimentParams.numSegmentsPerCurve,
                        m_pPerGlobalThreadRandStates,
                        m_pPerThreadPositions,
                        m_pPerThreadTangents,
                        m_pPerThreadCurvatures,
                        m_pPerThreadSegmentWeightCache,
                        m_pPerBlockFinalWeights,
                        m_pPerBlockFinalDifferences,
                        m_upInitialCurve->m_segmentLength,
                        m_experimentParams.weightingParameters.scatter,
                        m_experimentParams.weightingParameters.absorbtion,
                        m_pWeightLookupTable,
                        minCurvature,
                        maxCurvature,
                        curvatureStepSize);

                    CudaSafeErrorCheck(hipGetLastError(), "Perturb kernal launch error");
                    CudaSafeErrorCheck(hipDeviceSynchronize(), "Perturb kernel sync error");

                    std::cout << "Done with the perturb phase" << std::endl;
                }
            }
            auto perturbPhaseTimeEnd = std::chrono::high_resolution_clock::now();
            perturbTimeCount += std::chrono::duration_cast<std::chrono::milliseconds>(perturbPhaseTimeEnd - perturbPhaseTimeStart).count();
            /* ---------------------------- */

            //TODO: Read back weights
            /* ---------------------------- */
            auto weightCopyTimeStart = std::chrono::high_resolution_clock::now();
            std::vector<double> finalBlockWeights(perturbGridSize);
            std::vector<double> finalBlockDifferences(perturbGridSize);
            CudaSafeErrorCheck(hipMemcpy(finalBlockWeights.data(), m_pPerBlockFinalWeights, sizeof(double) * perturbGridSize, hipMemcpyDeviceToHost),
                "Copy back compressed weights from device");
            CudaSafeErrorCheck(hipMemcpy(finalBlockDifferences.data(), m_pPerBlockFinalDifferences, sizeof(double) * perturbGridSize, hipMemcpyDeviceToHost),
                "Copy back weight differences from device");
            auto weightCopyTimeEnd = std::chrono::high_resolution_clock::now();
            weightCopyTimeCount += std::chrono::duration_cast<std::chrono::milliseconds>(weightCopyTimeEnd - weightCopyTimeStart).count();
            /* ---------------------------- */


            // Temporarily print them out
            //std::cout << "Weights: " << std::endl;
            //for (uint32_t weightIdx = 0; weightIdx < 10; weightIdx++)
            //{
            //    std::cout << "Weight: " << compressedWeightBuffer[weightIdx] << std::endl;
            //}


            /* ---------------------------- */
            auto weightingTimeStart = std::chrono::high_resolution_clock::now();
            
            boost::multiprecision::cpp_dec_float_100 totalDispatchWeight = 0.0;
            {
                /*               std::vector<boost::multiprecision::cpp_dec_float_100> minimums(numPerturbThreads);
                                std::vector<boost::multiprecision::cpp_dec_float_100> maximums(numPerturbThreads);*/
                for (uint32_t threadIdx = 0; threadIdx < perturbGridSize; ++threadIdx)
                {
                    //std::cout << "Final thread weight " << threadIdx << ": " << finalThreadWeights[threadIdx] << std::endl;
                    //std::cout << "Final thread difference " << threadIdx << ": " << finalThreadDifferences[threadIdx] << std::endl;


                    boost::multiprecision::cpp_dec_float_100 bigFloatTotalWeight = finalBlockWeights[threadIdx];
                    boost::multiprecision::cpp_dec_float_100 bigFloatDifference = finalBlockDifferences[threadIdx];
                    boost::multiprecision::cpp_dec_float_100 log10BigFloatWeight = boost::multiprecision::log10(bigFloatTotalWeight);
                    boost::multiprecision::cpp_dec_float_100 adjustedLog10BigFloatWeight = log10BigFloatWeight - bigFloatDifference;
                    boost::multiprecision::cpp_dec_float_100 finalWeight = boost::multiprecision::pow(10, adjustedLog10BigFloatWeight);

                    // std::cout << "Thread " << threadIdx << ": " << finalWeight << ::std::endl;

                    totalDispatchWeight += finalWeight;
                }
            }

            // Full path normalization term
            // Hardcoded value from Jerry analysis.
            boost::multiprecision::cpp_dec_float_100 singleSegmentNormalizer = 2.0 * TwistyPi * boost::multiprecision::exp(boost::multiprecision::cpp_dec_float_100(0.625));
            boost::multiprecision::cpp_dec_float_100 segmentNormalizer = 1.0;
            for (uint32_t segIdx = 0; segIdx < (m_experimentParams.numSegmentsPerCurve - 1); ++segIdx)
            {
                segmentNormalizer = segmentNormalizer * singleSegmentNormalizer;
            }

            boost::multiprecision::cpp_dec_float_100 pathNormalizer = 1.0;
            pathNormalizer = pathNormalizer * boost::multiprecision::pow(boost::multiprecision::cpp_dec_float_100(static_cast<float>(m_experimentParams.numSegmentsPerCurve) / m_upInitialCurve->m_arclength), 3.0);
            pathNormalizer = pathNormalizer * segmentNormalizer;
            pathNormalizer = pathNormalizer * boost::multiprecision::exp(boost::multiprecision::cpp_dec_float_100(-0.325));
            totalDispatchWeight *= pathNormalizer;

            std::cout << "Final Weight: " << totalDispatchWeight << std::endl;

            bigTotalExperimentWeight += totalDispatchWeight;

            auto weightingTimeEnd = std::chrono::high_resolution_clock::now();
            weightCalcTimeCount += std::chrono::duration_cast<std::chrono::milliseconds>(weightingTimeEnd - weightingTimeStart).count();
            /* ---------------------------- */

            //numPathsLeft -= pathsInDispatch;
            //numPathsGenerated += pathsInDispatch;
        }

        // Now that they're read back, delete the allocated memory
        CleanupCudaMemory();

        auto runExperimentTimeEnd = std::chrono::high_resolution_clock::now();

        std::cout << "Experiment Time Reporting: " << std::endl;
        auto runExperimentTimeMs = std::chrono::duration_cast<std::chrono::milliseconds>(runExperimentTimeEnd - runExperimentTimeStart);
        std::cout << "\tTotal Experiment Time: " << runExperimentTimeMs.count() << "ms" << std::endl;

        {
            auto timeMs = std::chrono::duration_cast<std::chrono::milliseconds>(setupCurandTimeEnd - setupCurandTimeStart);
            std::cout << "\tsetupCurand Time: " << timeMs.count() << "ms - " << ((float)timeMs.count() / (float)runExperimentTimeMs.count()) * 100.0f << "%" << std::endl;
        }

        {
            auto timeMs = std::chrono::duration_cast<std::chrono::milliseconds>(setupCurveDataStructuresTimeEnd - setupCurveDataStructuresTimeStart);
            std::cout << "\tsetupCurveDataStructures Time: " << timeMs.count() << "ms - " << ((float)timeMs.count() / (float)runExperimentTimeMs.count()) * 100.0f << "%" << std::endl;
        }

        {
            std::cout << "\tperturbPhase Time: " << perturbTimeCount << "ms - " << ((float)perturbTimeCount / (float)runExperimentTimeMs.count()) * 100.0f << "%" << std::endl;
        }

        {
            std::cout << "\tweightCopy Time: " << weightCopyTimeCount << "ms - " << ((float)weightCopyTimeCount / (float)runExperimentTimeMs.count()) * 100.0f << "%" << std::endl;
        }

        {
            std::cout << "\tweighting Time: " << weightCalcTimeCount << "ms - " << ((float)weightCalcTimeCount / (float)runExperimentTimeMs.count()) * 100.0f << "%" << std::endl;
        }


        ExperimentResults results;
        results.experimentWeight = bigTotalExperimentWeight;
        results.totalPathsGenerated = m_experimentParams.numPathsInExperiment;
        results.numFailedPaths = 0;
        return results;
    }

    void GpuFullExperimentRunnerGeneral2::Shutdown()
    {
    }

    static std::pair<float, float> CurvatureAndTorsionBetweenTwoFrames(const Farlor::Matrix3x3& startFrame, const Farlor::Matrix3x3& endFrame, float segmentLength)
    {
        std::pair<float, float> curvatureAndTorsion = { 0.0f, 0.0f };
        {
            float curvature = ((endFrame.m_rows[0] - startFrame.m_rows[0]) * (1.0f / segmentLength)).Magnitude();
            curvatureAndTorsion.first = curvature;
        }

        {
            auto torsionLeft = -1.0f * startFrame.m_rows[1];
            auto torsionRight = (endFrame.m_rows[2] - startFrame.m_rows[2]) * (1.0f / segmentLength);
            float torsion = torsionLeft.Dot(torsionRight);
            curvatureAndTorsion.second = torsion;
        }
        return curvatureAndTorsion;
    }
}